#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdint>
#include <filesystem>
#include <memory>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_resources.hpp>
#include <raft/random/make_blobs.cuh>
#include <string>

#include <cuvs/neighbors/cagra.hpp>
#include <cuvs/neighbors/hnsw.hpp>

#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

#include "common.cuh"

void cagra_build_search_ace(raft::device_resources const& dev_resources,
                            raft::device_matrix_view<const float, int64_t> dataset,
                            raft::device_matrix_view<const float, int64_t> queries)
{
  using namespace cuvs::neighbors;

  int64_t topk      = 12;
  int64_t n_queries = queries.extent(0);

  // create output arrays
  auto neighbors = raft::make_device_matrix<uint32_t>(dev_resources, n_queries, topk);
  auto distances = raft::make_device_matrix<float>(dev_resources, n_queries, topk);

  // use ACE index parameters
  cagra::index_params index_params;
  auto ace_params            = cagra::graph_build_params::ace_params();
  ace_params.ace_npartitions = 4;
  ace_params.ace_build_dir   = "/tmp/ace_build";
  // ace_params.ace_use_disk         = true;  // Uncomment to use disk storage
  index_params.graph_build_params = ace_params;

  // ACE requires the dataset to be on the host
  auto dataset_host = raft::make_host_matrix<float, int64_t>(dataset.extent(0), dataset.extent(1));
  raft::copy(dataset_host.data_handle(),
             dataset.data_handle(),
             dataset.extent(0) * dataset.extent(1),
             raft::resource::get_cuda_stream(dev_resources));
  raft::resource::sync_stream(dev_resources);
  auto dataset_host_view = raft::make_host_matrix_view<const float, int64_t, raft::row_major>(
    dataset_host.data_handle(), dataset_host.extent(0), dataset_host.extent(1));

  std::cout << "Building CAGRA index (search graph)" << std::endl;
  auto index = cagra::build(dev_resources, index_params, dataset_host_view);

  std::cout << "CAGRA index has " << index.size() << " vectors" << std::endl;
  std::cout << "CAGRA graph has degree " << index.graph_degree() << ", graph size ["
            << index.graph().extent(0) << ", " << index.graph().extent(1) << "]" << std::endl;

  // use default search parameters
  cagra::search_params search_params;

  // Check if the partitioned build used disk storage
  if (index.on_disk()) {
    std::cout << "CAGRA index used disk storage. Create HNSW index from disk." << std::endl;

    hnsw::index_params hnsw_params;
    hnsw_params.hierarchy = hnsw::HnswHierarchy::GPU;

    auto hnsw_index = hnsw::from_cagra(dev_resources, hnsw_params, index);
    std::cout << "HNSW index serialized to disk. Deserializing..." << std::endl;

    // Deserialize the HNSW index from disk
    std::string hnsw_index_path        = index.file_directory() + "/hnsw_index.bin";
    hnsw::index<float>* hnsw_index_raw = nullptr;
    hnsw::deserialize(
      dev_resources, hnsw_params, hnsw_index_path, index.dim(), index.metric(), &hnsw_index_raw);

    std::unique_ptr<hnsw::index<float>> hnsw_index_deserialized(hnsw_index_raw);

    // HNSW search requires host matrices
    auto queries_host = raft::make_host_matrix<float, int64_t>(n_queries, queries.extent(1));
    raft::copy(queries_host.data_handle(),
               queries.data_handle(),
               n_queries * queries.extent(1),
               raft::resource::get_cuda_stream(dev_resources));
    raft::resource::sync_stream(dev_resources);

    // HNSW search outputs uint64_t indices
    auto indices_hnsw_host   = raft::make_host_matrix<uint64_t, int64_t>(n_queries, topk);
    auto distances_hnsw_host = raft::make_host_matrix<float, int64_t>(n_queries, topk);

    hnsw::search_params hnsw_search_params;
    hnsw_search_params.ef          = std::max(200, static_cast<int>(topk) * 2);
    hnsw_search_params.num_threads = 1;

    hnsw::search(dev_resources,
                 hnsw_search_params,
                 *hnsw_index_deserialized,
                 queries_host.view(),
                 indices_hnsw_host.view(),
                 distances_hnsw_host.view());

    // Convert uint64_t indices back to uint32_t and copy to device
    for (int64_t i = 0; i < n_queries * topk; i++) {
      neighbors.data_handle()[i] = static_cast<uint32_t>(indices_hnsw_host.data_handle()[i]);
    }
    raft::copy(distances.data_handle(),
               distances_hnsw_host.data_handle(),
               n_queries * topk,
               raft::resource::get_cuda_stream(dev_resources));
    raft::resource::sync_stream(dev_resources);
  } else {
    std::cout << "CAGRA index created in memory." << std::endl;

    // search K nearest neighbors
    cagra::search(dev_resources, search_params, index, queries, neighbors.view(), distances.view());
  }

  // The call to cagra::search is asynchronous. Before accessing the data, sync by calling
  // raft::resource::sync_stream(dev_resources);

  print_results(dev_resources, neighbors.view(), distances.view());
}

int main()
{
  raft::device_resources dev_resources;

  // Set pool memory resource with 1 GiB initial pool size. All allocations use the same pool.
  rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource> pool_mr(
    rmm::mr::get_current_device_resource(), 1024 * 1024 * 1024ull);
  rmm::mr::set_current_device_resource(&pool_mr);

  // Alternatively, one could define a pool allocator for temporary arrays (used within RAFT
  // algorithms). In that case only the internal arrays would use the pool, any other allocation
  // uses the default RMM memory resource. Here is how to change the workspace memory resource to
  // a pool with 2 GiB upper limit.
  // raft::resource::set_workspace_to_pool_resource(dev_resources, 2 * 1024 * 1024 * 1024ull);

  // Create input arrays.
  int64_t n_samples = 10000;
  int64_t n_dim     = 90;
  int64_t n_queries = 10;
  auto dataset      = raft::make_device_matrix<float, int64_t>(dev_resources, n_samples, n_dim);
  auto queries      = raft::make_device_matrix<float, int64_t>(dev_resources, n_queries, n_dim);
  generate_dataset(dev_resources, dataset.view(), queries.view());

  // ACE build and search example.
  cagra_build_search_ace(dev_resources,
                         raft::make_const_mdspan(dataset.view()),
                         raft::make_const_mdspan(queries.view()));
}

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"
#include "spatial_data.h"

#include <cuvs/distance/distance.hpp>
#include <cuvs/neighbors/ball_cover.hpp>
#include <cuvs/neighbors/brute_force.hpp>
#include <raft/core/device_mdspan.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resource/thrust_policy.hpp>
#include <raft/random/make_blobs.cuh>
#include <raft/util/cudart_utils.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/count.h>
#include <thrust/fill.h>
#include <thrust/transform.h>

#include <gtest/gtest.h>

#include <cstdint>
#include <iostream>
#include <vector>

namespace cuvs::neighbors::ball_cover {
using namespace std;

template <typename value_idx, typename value_t>
RAFT_KERNEL count_discrepancies_kernel(value_idx* actual_idx,
                                       value_idx* expected_idx,
                                       value_t* actual,
                                       value_t* expected,
                                       uint32_t m,
                                       uint32_t n,
                                       uint32_t* out,
                                       float thres = 1e-3)
{
  uint32_t row = blockDim.x * blockIdx.x + threadIdx.x;

  int n_diffs = 0;
  if (row < m) {
    for (uint32_t i = 0; i < n; i++) {
      value_t d    = actual[row * n + i] - expected[row * n + i];
      bool matches = (fabsf(d) <= thres) || (actual_idx[row * n + i] == expected_idx[row * n + i] &&
                                             actual_idx[row * n + i] == row);

      if (!matches) {
        printf(
          "row=%ud, n=%ud, actual_dist=%f, actual_ind=%ld, expected_dist=%f, expected_ind=%ld\n",
          row,
          i,
          actual[row * n + i],
          actual_idx[row * n + i],
          expected[row * n + i],
          expected_idx[row * n + i]);
      }
      n_diffs += !matches;
      out[row] = n_diffs;
    }
  }
}

struct is_nonzero {
  __host__ __device__ bool operator()(uint32_t i) { return i > 0; }
};

template <typename value_idx, typename value_t>
uint32_t count_discrepancies(value_idx* actual_idx,
                             value_idx* expected_idx,
                             value_t* actual,
                             value_t* expected,
                             uint32_t m,
                             uint32_t n,
                             uint32_t* out,
                             hipStream_t stream)
{
  uint32_t tpb = 256;
  count_discrepancies_kernel<<<raft::ceildiv(m, tpb), tpb, 0, stream>>>(
    actual_idx, expected_idx, actual, expected, m, n, out);

  // Copy results to host
  std::vector<uint32_t> h_out(m);
  raft::copy(h_out.data(), out, m, stream);
  RAFT_CUDA_TRY(hipStreamSynchronize(stream));

  // Count non-zero elements on the host
  uint32_t count = 0;
  for (uint32_t i = 0; i < m; i++) {
    if (h_out[i] > 0) count++;
  }

  return count;
}

template <typename value_t>
void compute_bfknn(const raft::resources& handle,
                   const value_t* X1,
                   const value_t* X2,
                   uint32_t n_rows,
                   uint32_t n_query_rows,
                   uint32_t d,
                   uint32_t k,
                   const cuvs::distance::DistanceType metric,
                   value_t* dists,
                   int64_t* inds)
{
  raft::device_matrix_view<const value_t, int64_t> input_vec =
    raft::make_device_matrix_view(X1, n_rows, d);
  auto bf_index_params   = cuvs::neighbors::brute_force::index_params();
  bf_index_params.metric = metric;
  auto bfindex           = cuvs::neighbors::brute_force::build(handle, bf_index_params, input_vec);

  auto bf_search_params = cuvs::neighbors::brute_force::search_params();
  cuvs::neighbors::brute_force::search(
    handle,
    bf_search_params,
    bfindex,
    raft::make_device_matrix_view<const value_t, int64_t>(X2, n_query_rows, d),
    raft::make_device_matrix_view<int64_t, int64_t>(inds, n_query_rows, k),
    raft::make_device_matrix_view<value_t, int64_t>(dists, n_query_rows, k));
}

struct ToRadians {
  __device__ __host__ float operator()(float a) { return a * (HIP_PI_F / 180.0); }
};

template <typename value_int = std::int64_t>
struct BallCoverInputs {
  value_int k;
  value_int n_rows;
  value_int n_cols;
  float weight;
  value_int n_query;
  cuvs::distance::DistanceType metric;
};

template <typename value_idx, typename value_t, typename value_int = std::int64_t>
class BallCoverKNNQueryTest : public ::testing::TestWithParam<BallCoverInputs<value_int>> {
 protected:
  void basicTest()
  {
    params = ::testing::TestWithParam<BallCoverInputs<value_int>>::GetParam();
    raft::resources handle;

    uint32_t k         = params.k;
    uint32_t n_centers = 25;
    float weight       = params.weight;
    auto metric        = params.metric;

    rmm::device_uvector<value_t> X(params.n_rows * params.n_cols,
                                   raft::resource::get_cuda_stream(handle));
    rmm::device_uvector<value_idx> Y(params.n_rows, raft::resource::get_cuda_stream(handle));

    // Make sure the train and query sets are completely disjoint
    rmm::device_uvector<value_t> X2(params.n_query * params.n_cols,
                                    raft::resource::get_cuda_stream(handle));
    rmm::device_uvector<value_idx> Y2(params.n_query, raft::resource::get_cuda_stream(handle));

    raft::random::make_blobs<value_t, value_idx>(X.data(),
                                                 Y.data(),
                                                 params.n_rows,
                                                 params.n_cols,
                                                 n_centers,
                                                 raft::resource::get_cuda_stream(handle));

    raft::random::make_blobs<value_t, value_idx>(X2.data(),
                                                 Y2.data(),
                                                 params.n_query,
                                                 params.n_cols,
                                                 n_centers,
                                                 raft::resource::get_cuda_stream(handle));

    rmm::device_uvector<value_idx> d_ref_I(params.n_query * k,
                                           raft::resource::get_cuda_stream(handle));
    rmm::device_uvector<value_t> d_ref_D(params.n_query * k,
                                         raft::resource::get_cuda_stream(handle));

    if (metric == cuvs::distance::DistanceType::Haversine) {
      thrust::transform(raft::resource::get_thrust_policy(handle),
                        X.data(),
                        X.data() + X.size(),
                        X.data(),
                        ToRadians());
      thrust::transform(raft::resource::get_thrust_policy(handle),
                        X2.data(),
                        X2.data() + X2.size(),
                        X2.data(),
                        ToRadians());
    }

    compute_bfknn(handle,
                  X.data(),
                  X2.data(),
                  params.n_rows,
                  params.n_query,
                  params.n_cols,
                  k,
                  metric,
                  d_ref_D.data(),
                  d_ref_I.data());

    raft::resource::sync_stream(handle);

    // Allocate predicted arrays
    rmm::device_uvector<value_idx> d_pred_I(params.n_query * k,
                                            raft::resource::get_cuda_stream(handle));
    rmm::device_uvector<value_t> d_pred_D(params.n_query * k,
                                          raft::resource::get_cuda_stream(handle));

    auto X_view =
      raft::make_device_matrix_view<value_t, value_int>(X.data(), params.n_rows, params.n_cols);
    auto X2_view = raft::make_device_matrix_view<const value_t, value_int>(
      (const value_t*)X2.data(), params.n_query, params.n_cols);

    auto d_pred_I_view =
      raft::make_device_matrix_view<value_idx, value_int>(d_pred_I.data(), params.n_query, k);
    auto d_pred_D_view =
      raft::make_device_matrix_view<value_t, value_int>(d_pred_D.data(), params.n_query, k);

    cuvs::neighbors::ball_cover::index<value_idx, value_t, value_int, value_int> index(
      handle, X_view, metric);
    cuvs::neighbors::ball_cover::build(handle, index);
    cuvs::neighbors::ball_cover::knn_query(
      handle, index, X2_view, d_pred_I_view, d_pred_D_view, k, true);

    raft::resource::sync_stream(handle);
    // What we really want are for the distances to match exactly. The
    // indices may or may not match exactly, depending upon the ordering which
    // can be nondeterministic.

    rmm::device_uvector<uint32_t> discrepancies(params.n_query,
                                                raft::resource::get_cuda_stream(handle));
    thrust::fill(raft::resource::get_thrust_policy(handle),
                 discrepancies.data(),
                 discrepancies.data() + discrepancies.size(),
                 0);
    //
    int res = count_discrepancies(d_ref_I.data(),
                                  d_pred_I.data(),
                                  d_ref_D.data(),
                                  d_pred_D.data(),
                                  params.n_query,
                                  k,
                                  discrepancies.data(),
                                  raft::resource::get_cuda_stream(handle));

    ASSERT_TRUE(res == 0);
  }

  void SetUp() override {}

  void TearDown() override {}

 protected:
  uint32_t d = 2;
  BallCoverInputs<value_int> params;
};

template <typename value_idx, typename value_t, typename value_int = std::int64_t>
class BallCoverAllKNNTest : public ::testing::TestWithParam<BallCoverInputs<value_int>> {
 protected:
  void basicTest()
  {
    params = ::testing::TestWithParam<BallCoverInputs<value_int>>::GetParam();
    raft::resources handle;

    uint32_t k         = params.k;
    uint32_t n_centers = 25;
    float weight       = params.weight;
    auto metric        = params.metric;

    rmm::device_uvector<value_t> X(params.n_rows * params.n_cols,
                                   raft::resource::get_cuda_stream(handle));
    rmm::device_uvector<value_int> Y(params.n_rows, raft::resource::get_cuda_stream(handle));

    raft::random::make_blobs<value_t, value_idx>(X.data(),
                                                 Y.data(),
                                                 params.n_rows,
                                                 params.n_cols,
                                                 n_centers,
                                                 raft::resource::get_cuda_stream(handle));

    rmm::device_uvector<value_idx> d_ref_I(params.n_rows * k,
                                           raft::resource::get_cuda_stream(handle));
    rmm::device_uvector<value_t> d_ref_D(params.n_rows * k,
                                         raft::resource::get_cuda_stream(handle));

    auto X_view = raft::make_device_matrix_view<const value_t, value_int>(
      (const value_t*)X.data(), params.n_rows, params.n_cols);

    if (metric == cuvs::distance::DistanceType::Haversine) {
      thrust::transform(raft::resource::get_thrust_policy(handle),
                        X.data(),
                        X.data() + X.size(),
                        X.data(),
                        ToRadians());
    }

    compute_bfknn(handle,
                  X.data(),
                  X.data(),
                  params.n_rows,
                  params.n_rows,
                  params.n_cols,
                  k,
                  metric,
                  d_ref_D.data(),
                  d_ref_I.data());

    raft::resource::sync_stream(handle);

    // Allocate predicted arrays
    rmm::device_uvector<value_idx> d_pred_I(params.n_rows * k,
                                            raft::resource::get_cuda_stream(handle));
    rmm::device_uvector<value_t> d_pred_D(params.n_rows * k,
                                          raft::resource::get_cuda_stream(handle));

    auto d_pred_I_view =
      raft::make_device_matrix_view<value_idx, value_int>(d_pred_I.data(), params.n_rows, k);
    auto d_pred_D_view =
      raft::make_device_matrix_view<value_t, value_int>(d_pred_D.data(), params.n_rows, k);

    cuvs::neighbors::ball_cover::index<value_idx, value_t> index(handle, X_view, metric);

    cuvs::neighbors::ball_cover::all_knn_query(
      handle, index, d_pred_I_view, d_pred_D_view, k, true);

    raft::resource::sync_stream(handle);
    // What we really want are for the distances to match exactly. The
    // indices may or may not match exactly, depending upon the ordering which
    // can be nondeterministic.

    rmm::device_uvector<uint32_t> discrepancies(params.n_rows,
                                                raft::resource::get_cuda_stream(handle));
    thrust::fill(raft::resource::get_thrust_policy(handle),
                 discrepancies.data(),
                 discrepancies.data() + discrepancies.size(),
                 0);
    //
    uint32_t res = count_discrepancies(d_ref_I.data(),
                                       d_pred_I.data(),
                                       d_ref_D.data(),
                                       d_pred_D.data(),
                                       params.n_rows,
                                       k,
                                       discrepancies.data(),
                                       raft::resource::get_cuda_stream(handle));

    // TODO: There seem to be discrepancies here only when
    // the entire test suite is executed.
    // Ref: https://github.com/rapidsai/raft/issues/
    // 1-5 mismatches in 8000 samples is 0.0125% - 0.0625%
    ASSERT_TRUE(res <= 5);
  }

  void SetUp() override {}

  void TearDown() override {}

 protected:
  BallCoverInputs<value_int> params;
};

typedef BallCoverAllKNNTest<int64_t, float> BallCoverAllKNNTestF;
typedef BallCoverKNNQueryTest<int64_t, float> BallCoverKNNQueryTestF;

const std::vector<BallCoverInputs<std::int64_t>> ballcover_inputs = {
  {11, 5000, 2, 1.0, 10000, cuvs::distance::DistanceType::Haversine},
  {25, 10000, 2, 1.0, 5000, cuvs::distance::DistanceType::Haversine},
  {2, 10000, 2, 1.0, 5000, cuvs::distance::DistanceType::L2SqrtUnexpanded},
  {2, 5000, 2, 1.0, 10000, cuvs::distance::DistanceType::Haversine},
  {11, 10000, 2, 1.0, 5000, cuvs::distance::DistanceType::L2SqrtUnexpanded},
  {25, 5000, 2, 1.0, 10000, cuvs::distance::DistanceType::L2SqrtUnexpanded},
  {5, 8000, 3, 1.0, 10000, cuvs::distance::DistanceType::L2SqrtUnexpanded},
  {11, 6000, 3, 1.0, 10000, cuvs::distance::DistanceType::L2SqrtUnexpanded},
  {25, 10000, 3, 1.0, 5000, cuvs::distance::DistanceType::L2SqrtUnexpanded}};

INSTANTIATE_TEST_CASE_P(BallCoverAllKNNTest,
                        BallCoverAllKNNTestF,
                        ::testing::ValuesIn(ballcover_inputs));
INSTANTIATE_TEST_CASE_P(BallCoverKNNQueryTest,
                        BallCoverKNNQueryTestF,
                        ::testing::ValuesIn(ballcover_inputs));

TEST_P(BallCoverAllKNNTestF, Fit) { basicTest(); }
TEST_P(BallCoverKNNQueryTestF, Fit) { basicTest(); }

}  // namespace cuvs::neighbors::ball_cover

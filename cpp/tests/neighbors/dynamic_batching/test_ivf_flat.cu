#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <gtest/gtest.h>

#include "../dynamic_batching.cuh"

#include <cuvs/neighbors/ivf_flat.hpp>

namespace cuvs::neighbors::dynamic_batching {

using ivf_flat_i8 = dynamic_batching_test<uint8_t,
                                          int64_t,
                                          ivf_flat::index<uint8_t, int64_t>,
                                          ivf_flat::build,
                                          ivf_flat::search>;

TEST_P(ivf_flat_i8, defaults)
{
  build_params_upsm.n_lists = std::round(std::sqrt(ps.n_rows));
  search_params_upsm.n_probes =
    std::max<uint32_t>(std::min<uint32_t>(build_params_upsm.n_lists, 10),
                       raft::div_rounding_up_safe<uint32_t>(build_params_upsm.n_lists, 50));
  build_all();
  search_all();
  check_neighbors();
}

INSTANTIATE_TEST_CASE_P(dynamic_batching, ivf_flat_i8, ::testing::ValuesIn(inputs));

}  // namespace cuvs::neighbors::dynamic_batching

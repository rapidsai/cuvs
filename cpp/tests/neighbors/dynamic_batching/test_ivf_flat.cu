#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include "../dynamic_batching.cuh"

#include <cuvs/neighbors/ivf_flat.hpp>

namespace cuvs::neighbors::dynamic_batching {

using ivf_flat_i8 = dynamic_batching_test<uint8_t,
                                          int64_t,
                                          ivf_flat::index<uint8_t, int64_t>,
                                          ivf_flat::build,
                                          ivf_flat::search>;

TEST_P(ivf_flat_i8, defaults)
{
  build_params_upsm.n_lists = std::round(std::sqrt(ps.n_rows));
  search_params_upsm.n_probes =
    std::max<uint32_t>(std::min<uint32_t>(build_params_upsm.n_lists, 10),
                       raft::div_rounding_up_safe<uint32_t>(build_params_upsm.n_lists, 50));
  build_all();
  search_all();
  check_neighbors();
}

INSTANTIATE_TEST_CASE_P(dynamic_batching, ivf_flat_i8, ::testing::ValuesIn(inputs));

}  // namespace cuvs::neighbors::dynamic_batching

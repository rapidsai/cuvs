#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuvs/neighbors/brute_force.hpp>
#include <cuvs/preprocessing/spectral_embedding.hpp>

#include <raft/core/device_coo_matrix.hpp>
#include <raft/core/device_mdspan.hpp>
#include <raft/core/handle.hpp>
#include <raft/core/resources.hpp>
#include <raft/linalg/matrix_vector_op.cuh>
#include <raft/matrix/gather.cuh>
#include <raft/matrix/init.cuh>
#include <raft/sparse/coo.hpp>
#include <raft/sparse/linalg/laplacian.cuh>
#include <raft/sparse/linalg/symmetrize.cuh>
#include <raft/sparse/op/filter.cuh>
#include <raft/sparse/solver/lanczos.cuh>
#include <raft/sparse/solver/lanczos_types.hpp>
#include <raft/util/cudart_utils.hpp>
#include <raft/util/integer_utils.hpp>

#include <thrust/sequence.h>
#include <thrust/tabulate.h>

namespace cuvs::preprocessing::spectral_embedding::helpers {

void create_connectivity_graph(
  raft::resources const& handle,
  cuvs::preprocessing::spectral_embedding::params spectral_embedding_config,
  raft::device_matrix_view<float, int, raft::row_major> dataset,
  raft::device_coo_matrix<float, int, int, int>& connectivity_graph)
{
  const int n_samples  = dataset.extent(0);
  const int n_features = dataset.extent(1);
  const int k_search   = spectral_embedding_config.n_neighbors;
  const size_t nnz     = n_samples * k_search;

  auto stream = raft::resource::get_cuda_stream(handle);

  cuvs::neighbors::brute_force::search_params search_params;
  cuvs::neighbors::brute_force::index_params index_params;
  index_params.metric = cuvs::distance::DistanceType::L2SqrtExpanded;

  auto d_indices   = raft::make_device_matrix<int64_t>(handle, n_samples, k_search);
  auto d_distances = raft::make_device_matrix<float>(handle, n_samples, k_search);

  auto index =
    cuvs::neighbors::brute_force::build(handle, index_params, raft::make_const_mdspan(dataset));

  cuvs::neighbors::brute_force::search(
    handle, search_params, index, dataset, d_indices.view(), d_distances.view());

  auto knn_rows = raft::make_device_vector<int>(handle, nnz);
  auto knn_cols = raft::make_device_vector<int>(handle, nnz);

  raft::linalg::unary_op(
    handle, make_const_mdspan(d_indices.view()), knn_cols.view(), [] __device__(int64_t x) {
      return static_cast<int>(x);
    });

  thrust::tabulate(raft::resource::get_thrust_policy(handle),
                   knn_rows.data_handle(),
                   knn_rows.data_handle() + nnz,
                   [k_search] __device__(int idx) { return idx / k_search; });

  // set all distances to 1.0f (connectivity KNN graph)
  raft::matrix::fill(handle, raft::make_device_vector_view(d_distances.data_handle(), nnz), 1.0f);

  auto coo_matrix_view = raft::make_device_coo_matrix_view<const float, int, int, int>(
    d_distances.data_handle(),
    raft::make_device_coordinate_structure_view<int, int, int>(
      knn_rows.data_handle(), knn_cols.data_handle(), n_samples, n_samples, nnz));

  auto sym_coo1_matrix =
    raft::make_device_coo_matrix<float, int, int, int>(handle, n_samples, n_samples);
  raft::sparse::linalg::coo_symmetrize<128, float, int, int>(
    handle, coo_matrix_view, sym_coo1_matrix, [] __device__(int row, int col, float a, float b) {
      return 0.5f * (a + b);
    });

  raft::sparse::matrix::set_diagonal(handle, sym_coo1_matrix.view(), static_cast<float>(0.0f));

  raft::sparse::op::coo_sort<float>(n_samples,
                                    n_samples,
                                    sym_coo1_matrix.structure_view().get_nnz(),
                                    sym_coo1_matrix.structure_view().get_rows().data(),
                                    sym_coo1_matrix.structure_view().get_cols().data(),
                                    sym_coo1_matrix.get_elements().data(),
                                    stream);

  raft::sparse::op::coo_remove_scalar<128, float, int, int>(
    handle,
    raft::make_device_coo_matrix_view<const float, int, int, int>(
      sym_coo1_matrix.get_elements().data(), sym_coo1_matrix.structure_view()),
    raft::make_host_scalar<float>(0.0f).view(),
    connectivity_graph);
}

}  // namespace cuvs::preprocessing::spectral_embedding::helpers

namespace cuvs::preprocessing::spectral_embedding {

raft::device_csr_matrix_view<float, int, int, int> coo_to_csr_matrix(
  raft::resources const& handle,
  const int n_samples,
  raft::device_vector_view<int> sym_coo_row_ind,
  raft::device_coo_matrix_view<float, int, int, int> sym_coo_matrix_view)
{
  auto stream = raft::resource::get_cuda_stream(handle);

  raft::sparse::op::coo_sort<float>(n_samples,
                                    n_samples,
                                    sym_coo_matrix_view.structure_view().get_nnz(),
                                    sym_coo_matrix_view.structure_view().get_rows().data(),
                                    sym_coo_matrix_view.structure_view().get_cols().data(),
                                    sym_coo_matrix_view.get_elements().data(),
                                    stream);

  raft::sparse::convert::sorted_coo_to_csr(sym_coo_matrix_view.structure_view().get_rows().data(),
                                           sym_coo_matrix_view.structure_view().get_nnz(),
                                           sym_coo_row_ind.data_handle(),
                                           n_samples,
                                           stream);

  auto sym_coo_nnz = sym_coo_matrix_view.structure_view().get_nnz();
  raft::copy(sym_coo_row_ind.data_handle() + sym_coo_row_ind.size() - 1, &sym_coo_nnz, 1, stream);

  auto csr_matrix_view = raft::make_device_csr_matrix_view<float, int, int, int>(
    const_cast<float*>(sym_coo_matrix_view.get_elements().data()),
    raft::make_device_compressed_structure_view<int, int, int>(
      const_cast<int*>(sym_coo_row_ind.data_handle()),
      const_cast<int*>(sym_coo_matrix_view.structure_view().get_cols().data()),
      n_samples,
      n_samples,
      sym_coo_matrix_view.structure_view().get_nnz()));
  return csr_matrix_view;
}

raft::device_csr_matrix<float, int, int, int> create_laplacian(
  raft::resources const& handle,
  params spectral_embedding_config,
  raft::device_csr_matrix_view<float, int, int, int> csr_matrix_view,
  raft::device_vector_view<float, int> diagonal)
{
  auto laplacian = spectral_embedding_config.norm_laplacian
                     ? raft::sparse::linalg::laplacian_normalized(handle, csr_matrix_view, diagonal)
                     : raft::sparse::linalg::compute_graph_laplacian(handle, csr_matrix_view);

  auto laplacian_elements_view = raft::make_device_vector_view<float, int>(
    laplacian.get_elements().data(), laplacian.structure_view().get_nnz());

  raft::linalg::unary_op(handle,
                         raft::make_const_mdspan(laplacian_elements_view),
                         laplacian_elements_view,
                         [] __device__(float x) { return -x; });

  return laplacian;
}

void compute_eigenpairs(raft::resources const& handle,
                        params spectral_embedding_config,
                        const int n_samples,
                        raft::device_csr_matrix<float, int, int, int>& laplacian,
                        raft::device_vector_view<float, int> diagonal,
                        raft::device_matrix_view<float, int, raft::col_major> embedding)
{
  auto config           = raft::sparse::solver::lanczos_solver_config<float>();
  config.n_components   = spectral_embedding_config.n_components;
  config.max_iterations = 10 * n_samples;
  config.ncv            = std::min(n_samples, std::max(2 * config.n_components + 1, 20));
  config.tolerance      = 1e-8;
  config.which          = raft::sparse::solver::LANCZOS_WHICH::LA;
  config.seed           = spectral_embedding_config.seed;

  auto eigenvalues =
    raft::make_device_vector<float, int, raft::col_major>(handle, config.n_components);
  auto eigenvectors =
    raft::make_device_matrix<float, int, raft::col_major>(handle, n_samples, config.n_components);

  raft::sparse::solver::lanczos_compute_smallest_eigenvectors<int, float>(
    handle,
    config,
    raft::make_device_csr_matrix_view<float, int, int, int>(laplacian.get_elements().data(),
                                                            laplacian.structure_view()),
    std::nullopt,
    eigenvalues.view(),
    eigenvectors.view());

  if (spectral_embedding_config.norm_laplacian) {
    raft::linalg::matrix_vector_op<raft::Apply::ALONG_COLUMNS>(
      handle,
      raft::make_const_mdspan(eigenvectors.view()),  // input matrix view
      raft::make_const_mdspan(diagonal),             // input vector view
      eigenvectors.view(),                           // output matrix view (in-place)
      [] __device__(float elem, float diag) { return elem / diag; });
  }

  // Create a sequence of reversed column indices
  config.n_components =
    spectral_embedding_config.drop_first ? config.n_components - 1 : config.n_components;
  auto col_indices = raft::make_device_vector<int>(handle, config.n_components);

  // TODO: https://github.com/rapidsai/raft/issues/2661
  thrust::sequence(thrust::device,
                   col_indices.data_handle(),
                   col_indices.data_handle() + config.n_components,
                   config.n_components - 1,  // Start from the last column index
                   -1                        // Decrement (move backward)
  );

  // Create row-major views of the column-major matrices
  // This is just a view re-interpretation, no data movement
  auto eigenvectors_row_view = raft::make_device_matrix_view<float, int, raft::row_major>(
    eigenvectors.data_handle(),
    eigenvectors.extent(1),  // Swap dimensions for the view
    eigenvectors.extent(0));

  auto embedding_row_view = raft::make_device_matrix_view<float, int, raft::row_major>(
    embedding.data_handle(),
    embedding.extent(1),  // Swap dimensions for the view
    embedding.extent(0));

  raft::matrix::gather<float, int, int>(
    handle,
    raft::make_const_mdspan(eigenvectors_row_view),  // Source matrix (as row-major view)
    raft::make_const_mdspan(col_indices.view()),     // Column indices to gather
    embedding_row_view                               // Destination matrix (as row-major view)
  );
}

void transform(raft::resources const& handle,
               params spectral_embedding_config,
               raft::device_matrix_view<float, int, raft::row_major> dataset,
               raft::device_matrix_view<float, int, raft::col_major> embedding)
{
  const int n_samples = dataset.extent(0);

  auto sym_coo_matrix =
    raft::make_device_coo_matrix<float, int, int, int>(handle, n_samples, n_samples);
  auto sym_coo_row_ind = raft::make_device_vector<int>(handle, n_samples + 1);
  auto diagonal        = raft::make_device_vector<float, int>(handle, n_samples);

  helpers::create_connectivity_graph(handle, spectral_embedding_config, dataset, sym_coo_matrix);
  auto csr_matrix_view =
    coo_to_csr_matrix(handle, n_samples, sym_coo_row_ind.view(), sym_coo_matrix.view());
  auto laplacian =
    create_laplacian(handle, spectral_embedding_config, csr_matrix_view, diagonal.view());
  compute_eigenpairs(
    handle, spectral_embedding_config, n_samples, laplacian, diagonal.view(), embedding);
}

void transform(raft::resources const& handle,
               params spectral_embedding_config,
               raft::device_coo_matrix_view<float, int, int, int> connectivity_graph,
               raft::device_matrix_view<float, int, raft::col_major> embedding)
{
  const int n_samples = connectivity_graph.structure_view().get_n_rows();

  auto sym_coo_row_ind = raft::make_device_vector<int>(handle, n_samples + 1);
  auto diagonal        = raft::make_device_vector<float, int>(handle, n_samples);

  auto csr_matrix_view =
    coo_to_csr_matrix(handle, n_samples, sym_coo_row_ind.view(), connectivity_graph);
  auto laplacian =
    create_laplacian(handle, spectral_embedding_config, csr_matrix_view, diagonal.view());
  compute_eigenpairs(
    handle, spectral_embedding_config, n_samples, laplacian, diagonal.view(), embedding);
}

}  // namespace cuvs::preprocessing::spectral_embedding

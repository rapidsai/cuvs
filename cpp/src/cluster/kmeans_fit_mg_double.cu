/*
 * SPDX-FileCopyrightText: Copyright (c) 2024, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */

#include "./detail/kmeans_mg.cuh"
#include "kmeans_mg.hpp"
#include <raft/core/resources.hpp>

namespace cuvs::cluster::kmeans::mg {

void fit(raft::resources const& handle,
         const cuvs::cluster::kmeans::params& params,
         raft::device_matrix_view<const double, int> X,
         std::optional<raft::device_vector_view<const double, int>> sample_weight,
         raft::device_matrix_view<double, int> centroids,
         raft::host_scalar_view<double, int> inertia,
         raft::host_scalar_view<int, int> n_iter)
{
  rmm::device_uvector<char> workspace(0, raft::resource::get_cuda_stream(handle));

  cuvs::cluster::kmeans::mg::detail::fit<double, int>(
    handle, params, X, sample_weight, centroids, inertia, n_iter, workspace);
}

void fit(raft::resources const& handle,
         const cuvs::cluster::kmeans::params& params,
         raft::device_matrix_view<const double, int64_t> X,
         std::optional<raft::device_vector_view<const double, int64_t>> sample_weight,
         raft::device_matrix_view<double, int64_t> centroids,
         raft::host_scalar_view<double, int64_t> inertia,
         raft::host_scalar_view<int64_t, int64_t> n_iter)
{
  rmm::device_uvector<char> workspace(0, raft::resource::get_cuda_stream(handle));

  cuvs::cluster::kmeans::mg::detail::fit<double, int64_t>(
    handle, params, X, sample_weight, centroids, inertia, n_iter, workspace);
}
}  // namespace cuvs::cluster::kmeans::mg

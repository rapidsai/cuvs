/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kmeans.cuh"
#include <raft/core/resources.hpp>

namespace cuvs::cluster::kmeans {

void fit(raft::resources const& handle,
         const cuvs::cluster::kmeans::params& params,
         raft::device_matrix_view<const double, int> X,
         std::optional<raft::device_vector_view<const double, int>> sample_weight,
         raft::device_matrix_view<double, int> centroids,
         raft::host_scalar_view<double, int> inertia,
         raft::host_scalar_view<int, int> n_iter)
{
  cuvs::cluster::kmeans::fit<double, int>(
    handle, params, X, sample_weight, centroids, inertia, n_iter);
}

void fit(raft::resources const& handle,
         const cuvs::cluster::kmeans::params& params,
         raft::device_matrix_view<const double, int64_t> X,
         std::optional<raft::device_vector_view<const double, int64_t>> sample_weight,
         raft::device_matrix_view<double, int64_t> centroids,
         raft::host_scalar_view<double, int64_t> inertia,
         raft::host_scalar_view<int64_t, int64_t> n_iter)
{
  cuvs::cluster::kmeans::fit<double, int64_t>(
    handle, params, X, sample_weight, centroids, inertia, n_iter);
}
}  // namespace cuvs::cluster::kmeans

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "detail/nn_descent.cuh"
#include <cuvs/neighbors/nn_descent.hpp>
#include <raft/core/logger.hpp>

using namespace raft;
namespace cuvs::neighbors::nn_descent {

/**
 * @brief Test if we have enough GPU memory to run NN descent algorithm.
 * *
 * @param res
 * @param dataset shape of the dataset
 * @param idx_size the size of index type in bytes
 * @return true if enough GPU memory could be allocated
 * @return false otherwise
 */
bool has_enough_device_memory(raft::resources const& res,
                              raft::matrix_extent<int64_t> dataset,
                              size_t idx_size)
{
  using DistData_t = float;
  try {
    auto d_data_ = raft::make_device_matrix<__half, size_t, raft::row_major>(
      res, dataset.extent(0), dataset.extent(1));
    auto l2_norms_     = raft::make_device_vector<DistData_t, size_t>(res, dataset.extent(0));
    auto graph_buffer_ = raft::make_device_vector<uint32_t, size_t>(
      res, dataset.extent(0) * idx_size * detail::DEGREE_ON_DEVICE);

    auto dists_buffer_ = raft::make_device_matrix<DistData_t, size_t, raft::row_major>(
      res, dataset.extent(0), detail::DEGREE_ON_DEVICE);

    auto d_locks_ = raft::make_device_vector<int, size_t>(res, dataset.extent(0));

    auto d_list_sizes_new_ = raft::make_device_vector<int2, size_t>(res, dataset.extent(0));
    auto d_list_sizes_old_ = raft::make_device_vector<int2, size_t>(res, dataset.extent(0));
    RAFT_LOG_DEBUG("Sufficient memory for NN descent");
    return true;
  } catch (std::bad_alloc& e) {
    RAFT_LOG_DEBUG("Insufficient memory for NN descent");
    return false;
  } catch (raft::logic_error& e) {
    RAFT_LOG_DEBUG("Insufficient memory for NN descent (logic error)");
    return false;
  }
}

}  // namespace cuvs::neighbors::nn_descent

#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../../core/nvtx.hpp"
#include "ivf_pq_build.cuh"
#include <cuvs/distance/distance.hpp>
#include <cuvs/neighbors/common.hpp>
#include <cuvs/neighbors/ivf_pq.hpp>
#include <raft/core/mdspan_types.hpp>

namespace cuvs::neighbors::ivf_pq::helpers {

namespace codepacker {

void unpack(
  raft::resources const& res,
  raft::device_mdspan<const uint8_t, list_spec<uint32_t, uint32_t>::list_extents, raft::row_major>
    list_data,
  uint32_t pq_bits,
  uint32_t offset,
  raft::device_matrix_view<uint8_t, uint32_t, raft::row_major> codes)
{
  detail::unpack_list_data(codes, list_data, offset, pq_bits, raft::resource::get_cuda_stream(res));
}

void unpack_contiguous(
  raft::resources const& res,
  raft::device_mdspan<const uint8_t, list_spec<uint32_t, uint32_t>::list_extents, raft::row_major>
    list_data,
  uint32_t pq_bits,
  uint32_t offset,
  uint32_t n_rows,
  uint32_t pq_dim,
  uint8_t* codes)
{
  detail::unpack_contiguous_list_data(
    codes, list_data, n_rows, pq_dim, offset, pq_bits, raft::resource::get_cuda_stream(res));
}
void pack(raft::resources const& res,
          raft::device_matrix_view<const uint8_t, uint32_t, raft::row_major> codes,
          uint32_t pq_bits,
          uint32_t offset,
          raft::device_mdspan<uint8_t, list_spec<uint32_t, uint32_t>::list_extents, raft::row_major>
            list_data)
{
  detail::pack_list_data(list_data, codes, offset, pq_bits, raft::resource::get_cuda_stream(res));
}

void pack_contiguous(
  raft::resources const& res,
  const uint8_t* codes,
  uint32_t n_rows,
  uint32_t pq_dim,
  uint32_t pq_bits,
  uint32_t offset,
  raft::device_mdspan<uint8_t, list_spec<uint32_t, uint32_t>::list_extents, raft::row_major>
    list_data)
{
  detail::pack_contiguous_list_data(
    list_data, codes, n_rows, pq_dim, offset, pq_bits, raft::resource::get_cuda_stream(res));
}

void pack_list_data(raft::resources const& res,
                    index<int64_t>* index,
                    raft::device_matrix_view<const uint8_t, uint32_t, raft::row_major> codes,
                    uint32_t label,
                    uint32_t offset)
{
  detail::pack_list_data(res, index, codes, label, offset);
}

void pack_contiguous_list_data(raft::resources const& res,
                               index<int64_t>* index,
                               uint8_t* codes,
                               uint32_t n_rows,
                               uint32_t label,
                               uint32_t offset)
{
  detail::pack_contiguous_list_data(res, index, codes, n_rows, label, offset);
}

void unpack_list_data(raft::resources const& res,
                      const index<int64_t>& index,
                      raft::device_matrix_view<uint8_t, uint32_t, raft::row_major> out_codes,
                      uint32_t label,
                      uint32_t offset)
{
  detail::unpack_list_data(res, index, out_codes, label, offset);
}

void unpack_list_data(raft::resources const& res,
                      const index<int64_t>& index,
                      raft::device_vector_view<const uint32_t> in_cluster_indices,
                      raft::device_matrix_view<uint8_t, uint32_t, raft::row_major> out_codes,
                      uint32_t label)
{
  detail::unpack_list_data<int64_t>(res, index, out_codes, label, in_cluster_indices.data_handle());
}

void unpack_contiguous_list_data(raft::resources const& res,
                                 const index<int64_t>& index,
                                 uint8_t* out_codes,
                                 uint32_t n_rows,
                                 uint32_t label,
                                 uint32_t offset)
{
  detail::unpack_contiguous_list_data(res, index, out_codes, n_rows, label, offset);
}

void reconstruct_list_data(raft::resources const& res,
                           const index<int64_t>& index,
                           raft::device_matrix_view<float, uint32_t, raft::row_major> out_vectors,
                           uint32_t label,
                           uint32_t offset)
{
  detail::reconstruct_list_data<float, int64_t>(res, index, out_vectors, label, offset);
}

void reconstruct_list_data(raft::resources const& res,
                           const index<int64_t>& index,
                           raft::device_matrix_view<int8_t, uint32_t, raft::row_major> out_vectors,
                           uint32_t label,
                           uint32_t offset)
{
  detail::reconstruct_list_data<int8_t, int64_t>(res, index, out_vectors, label, offset);
}
void reconstruct_list_data(raft::resources const& res,
                           const index<int64_t>& index,
                           raft::device_matrix_view<uint8_t, uint32_t, raft::row_major> out_vectors,
                           uint32_t label,
                           uint32_t offset)
{
  detail::reconstruct_list_data<uint8_t, int64_t>(res, index, out_vectors, label, offset);
}

void reconstruct_list_data(raft::resources const& res,
                           const index<int64_t>& index,
                           raft::device_vector_view<const uint32_t> in_cluster_indices,
                           raft::device_matrix_view<float, uint32_t, raft::row_major> out_vectors,
                           uint32_t label)
{
  detail::reconstruct_list_data<float, int64_t>(
    res, index, out_vectors, label, in_cluster_indices.data_handle());
}
void reconstruct_list_data(raft::resources const& res,
                           const index<int64_t>& index,
                           raft::device_vector_view<const uint32_t> in_cluster_indices,
                           raft::device_matrix_view<int8_t, uint32_t, raft::row_major> out_vectors,
                           uint32_t label)
{
  detail::reconstruct_list_data<int8_t, int64_t>(
    res, index, out_vectors, label, in_cluster_indices.data_handle());
}
void reconstruct_list_data(raft::resources const& res,
                           const index<int64_t>& index,
                           raft::device_vector_view<const uint32_t> in_cluster_indices,
                           raft::device_matrix_view<uint8_t, uint32_t, raft::row_major> out_vectors,
                           uint32_t label)
{
  detail::reconstruct_list_data<uint8_t, int64_t>(
    res, index, out_vectors, label, in_cluster_indices.data_handle());
}

void extend_list_with_codes(
  raft::resources const& res,
  index<int64_t>* index,
  raft::device_matrix_view<const uint8_t, uint32_t, raft::row_major> new_codes,
  raft::device_vector_view<const int64_t, uint32_t, raft::row_major> new_indices,
  uint32_t label)
{
  detail::extend_list_with_codes<int64_t>(res, index, new_codes, new_indices, label);
}

void extend_list(raft::resources const& res,
                 index<int64_t>* index,
                 raft::device_matrix_view<const float, uint32_t, raft::row_major> new_vectors,
                 raft::device_vector_view<const int64_t, uint32_t, raft::row_major> new_indices,
                 uint32_t label)
{
  detail::extend_list<float, int64_t>(res, index, new_vectors, new_indices, label);
}
void extend_list(raft::resources const& res,
                 index<int64_t>* index,
                 raft::device_matrix_view<const int8_t, uint32_t, raft::row_major> new_vectors,
                 raft::device_vector_view<const int64_t, uint32_t, raft::row_major> new_indices,
                 uint32_t label)
{
  detail::extend_list<int8_t, int64_t>(res, index, new_vectors, new_indices, label);
}
void extend_list(raft::resources const& res,
                 index<int64_t>* index,
                 raft::device_matrix_view<const uint8_t, uint32_t, raft::row_major> new_vectors,
                 raft::device_vector_view<const int64_t, uint32_t, raft::row_major> new_indices,
                 uint32_t label)
{
  detail::extend_list<uint8_t, int64_t>(res, index, new_vectors, new_indices, label);
}

};  // namespace codepacker

void erase_list(raft::resources const& res, index<int64_t>* index, uint32_t label)
{
  detail::erase_list<int64_t>(res, index, label);
}

void reset_index(const raft::resources& res, index<int64_t>* index)
{
  auto stream = raft::resource::get_cuda_stream(res);

  cuvs::spatial::knn::detail::utils::memzero(
    index->accum_sorted_sizes().data_handle(), index->accum_sorted_sizes().size(), stream);
  cuvs::spatial::knn::detail::utils::memzero(
    index->list_sizes().data_handle(), index->list_sizes().size(), stream);
  cuvs::spatial::knn::detail::utils::memzero(
    index->data_ptrs().data_handle(), index->data_ptrs().size(), stream);
  cuvs::spatial::knn::detail::utils::memzero(
    index->inds_ptrs().data_handle(), index->inds_ptrs().size(), stream);
}

void make_rotation_matrix(raft::resources const& handle,
                          bool force_random_rotation,
                          uint32_t n_rows,
                          uint32_t n_cols,
                          float* rotation_matrix,
                          raft::random::RngState rng = raft::random::RngState(7ULL))
{
  raft::common::nvtx::range<cuvs::common::nvtx::domain::cuvs> fun_scope(
    "ivf_pq::make_rotation_matrix(%u * %u)", n_rows, n_cols);
  auto stream  = raft::resource::get_cuda_stream(handle);
  bool inplace = n_rows == n_cols;
  uint32_t n   = std::max(n_rows, n_cols);
  if (force_random_rotation || !inplace) {
    rmm::device_uvector<float> buf(inplace ? 0 : n * n, stream);
    float* mat = inplace ? rotation_matrix : buf.data();
    raft::random::normal(handle, rng, mat, n * n, 0.0f, 1.0f);
    raft::linalg::detail::qrGetQ_inplace(handle, mat, n, n, stream);
    if (!inplace) {
      RAFT_CUDA_TRY(hipMemcpy2DAsync(rotation_matrix,
                                      sizeof(float) * n_cols,
                                      mat,
                                      sizeof(float) * n,
                                      sizeof(float) * n_cols,
                                      n_rows,
                                      hipMemcpyDefault,
                                      stream));
    }
  } else {
    uint32_t stride = n + 1;
    auto rotation_matrix_view =
      raft::make_device_vector_view<float, uint32_t>(rotation_matrix, n * n);
    raft::linalg::map_offset(handle, rotation_matrix_view, [stride] __device__(uint32_t i) {
      return static_cast<float>(i % stride == 0u);
    });
  }
}

void make_rotation_matrix(raft::resources const& res,
                          index<int64_t>* index,
                          bool force_random_rotation)
{
  make_rotation_matrix(res,
                       force_random_rotation,
                       index->rot_dim(),
                       index->dim(),
                       index->rotation_matrix().data_handle());
}

void set_centers(raft::resources const& handle,
                 index<int64_t>* index,
                 raft::device_matrix_view<const float, uint32_t, raft::row_major> cluster_centers)
{
  RAFT_EXPECTS(cluster_centers.extent(0) == index->n_lists(),
               "Number of rows in the new centers must be equal to the number of IVF lists");
  RAFT_EXPECTS(cluster_centers.extent(1) == index->dim(),
               "Number of columns in the new cluster centers and index dim are different");
  RAFT_EXPECTS(index->size() == 0, "Index must be empty");
  detail::set_centers(handle, index, cluster_centers.data_handle());
}

void extract_centers(raft::resources const& res,
                     const cuvs::neighbors::ivf_pq::index<int64_t>& index,
                     raft::device_matrix_view<float, uint32_t, raft::row_major> cluster_centers)
{
  RAFT_EXPECTS(cluster_centers.extent(0) == index.n_lists(),
               "Number of rows in the output buffer for cluster centers must be equal to the "
               "number of IVF lists");
  RAFT_EXPECTS(
    cluster_centers.extent(1) == index.dim(),
    "Number of columns in the output buffer for cluster centers and index dim are different");
  auto stream = raft::resource::get_cuda_stream(res);
  RAFT_CUDA_TRY(hipMemcpy2DAsync(cluster_centers.data_handle(),
                                  sizeof(float) * index.dim(),
                                  index.centers().data_handle(),
                                  sizeof(float) * index.dim_ext(),
                                  sizeof(float) * index.dim(),
                                  index.n_lists(),
                                  hipMemcpyDefault,
                                  stream));
}

void recompute_internal_state(const raft::resources& res, index<int64_t>* index)
{
  ivf::detail::recompute_internal_state(res, *index);
}

}  // namespace cuvs::neighbors::ivf_pq::helpers

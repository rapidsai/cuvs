/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cagra.cuh"
#include <cuvs/neighbors/cagra.hpp>

namespace cuvs::neighbors::cagra {

#define RAFT_INST_CAGRA_MERGE(T, IdxT)                                      \
  auto merge(raft::resources const& handle,                                 \
             const cuvs::neighbors::cagra::merge_params& params,            \
             std::vector<cuvs::neighbors::cagra::index<T, IdxT>*>& indices) \
    ->cuvs::neighbors::cagra::index<T, IdxT>                                \
  {                                                                         \
    return cuvs::neighbors::cagra::merge<T, IdxT>(handle, params, indices); \
  }

RAFT_INST_CAGRA_MERGE(float, uint32_t);

#undef RAFT_INST_CAGRA_MERGE

}  // namespace cuvs::neighbors::cagra

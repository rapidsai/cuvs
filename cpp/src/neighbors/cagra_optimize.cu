/*
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */

#include "cagra.cuh"
#include <cuvs/neighbors/cagra_optimize.hpp>

namespace cuvs::neighbors::cagra::helpers {

void optimize(raft::resources const& handle,
              raft::host_matrix_view<uint32_t, int64_t, raft::row_major> knn_graph,
              raft::host_matrix_view<uint32_t, int64_t, raft::row_major> new_graph)
{
  cuvs::neighbors::cagra::optimize(handle, knn_graph, new_graph);
}

}  // namespace cuvs::neighbors::cagra::helpers

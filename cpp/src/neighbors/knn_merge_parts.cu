/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "detail/knn_merge_parts.cuh"

#include <cuvs/neighbors/knn_merge_parts.hpp>

namespace cuvs::neighbors {
namespace {
template <typename T, typename IdxT>
void _knn_merge_parts(raft::resources const& res,
                      raft::device_matrix_view<const T, int64_t> inK,
                      raft::device_matrix_view<const IdxT, int64_t> inV,
                      raft::device_matrix_view<T, int64_t> outK,
                      raft::device_matrix_view<IdxT, int64_t> outV,
                      raft::device_vector_view<IdxT> translations)
{
  auto rows  = inK.extent(0);
  auto parts = translations.extent(0);
  auto k     = outK.extent(1);

  detail::knn_merge_parts(inK.data_handle(),
                          inV.data_handle(),
                          outK.data_handle(),
                          outV.data_handle(),
                          rows,
                          parts,
                          k,
                          raft::resource::get_cuda_stream(res),
                          translations.data_handle());
}
}  // namespace

void knn_merge_parts(raft::resources const& res,
                     raft::device_matrix_view<const float, int64_t> inK,
                     raft::device_matrix_view<const int64_t, int64_t> inV,
                     raft::device_matrix_view<float, int64_t> outK,
                     raft::device_matrix_view<int64_t, int64_t> outV,
                     raft::device_vector_view<int64_t> translations)
{
  _knn_merge_parts(res, inK, inV, outK, outV, translations);
}
void knn_merge_parts(raft::resources const& res,
                     raft::device_matrix_view<const float, int64_t> inK,
                     raft::device_matrix_view<const uint32_t, int64_t> inV,
                     raft::device_matrix_view<float, int64_t> outK,
                     raft::device_matrix_view<uint32_t, int64_t> outV,
                     raft::device_vector_view<uint32_t> translations)
{
  _knn_merge_parts(res, inK, inV, outK, outV, translations);
}
void knn_merge_parts(raft::resources const& res,
                     raft::device_matrix_view<const float, int64_t> inK,
                     raft::device_matrix_view<const int32_t, int64_t> inV,
                     raft::device_matrix_view<float, int64_t> outK,
                     raft::device_matrix_view<int32_t, int64_t> outV,
                     raft::device_vector_view<int32_t> translations)
{
  _knn_merge_parts(res, inK, inV, outK, outV, translations);
}
}  // namespace cuvs::neighbors

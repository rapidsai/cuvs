/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "vamana.cuh"
#include <cuvs/neighbors/vamana.hpp>

namespace cuvs::neighbors::experimental::vamana {

#define RAFT_INST_VAMANA_BUILD(T, IdxT)                                                    \
  auto build(raft::resources const& handle,                                                \
             const cuvs::neighbors::experimental::vamana::index_params& params,            \
             raft::device_matrix_view<const T, int64_t, raft::row_major> dataset)          \
    ->cuvs::neighbors::experimental::vamana::index<T, IdxT>                                \
  {                                                                                        \
    return cuvs::neighbors::experimental::vamana::build<T, IdxT>(handle, params, dataset); \
  }                                                                                        \
                                                                                           \
  auto build(raft::resources const& handle,                                                \
             const cuvs::neighbors::experimental::vamana::index_params& params,            \
             raft::host_matrix_view<const T, int64_t, raft::row_major> dataset)            \
    ->cuvs::neighbors::experimental::vamana::index<T, IdxT>                                \
  {                                                                                        \
    return cuvs::neighbors::experimental::vamana::build<T, IdxT>(handle, params, dataset); \
  }

RAFT_INST_VAMANA_BUILD(float, uint32_t);

#undef RAFT_INST_VAMANA_BUILD

}  // namespace cuvs::neighbors::experimental::vamana

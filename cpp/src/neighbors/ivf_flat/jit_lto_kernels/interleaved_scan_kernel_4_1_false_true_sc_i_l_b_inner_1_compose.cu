#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../ivf_flat_interleaved_scan.cuh"

#ifdef BUILD_KERNEL

template __global__ void cuvs::neighbors::ivf_flat::detail::interleaved_scan_kernel<
  4,
  1,
  false,
  true,
  signed char,
  int,
  long,
  cuvs::neighbors::filtering::
    ivf_to_sample_filter<long, cuvs::neighbors::filtering::bitset_filter<unsigned int, long>>,
  cuvs::neighbors::ivf_flat::detail::inner_prod_dist<1, signed char, int>,
  raft::compose_op<raft::plug_const_op<float, raft::add_op>,
                   raft::plug_const_op<float, raft::mul_op>>>(
  cuvs::neighbors::ivf_flat::detail::inner_prod_dist<1, signed char, int>,
  raft::compose_op<raft::plug_const_op<float, raft::add_op>,
                   raft::plug_const_op<float, raft::mul_op>>,
  unsigned int,
  signed char const*,
  unsigned int const*,
  signed char const* const*,
  unsigned int const*,
  unsigned int,
  unsigned int,
  unsigned int,
  unsigned int,
  unsigned int const*,
  unsigned int,
  cuvs::neighbors::filtering::
    ivf_to_sample_filter<long, cuvs::neighbors::filtering::bitset_filter<unsigned int, long>>,
  unsigned int*,
  float*);

#else

#include "interleaved_scan_kernel_4_1_false_true_sc_i_l_b_inner_1_compose.h"
#include <cuvs/detail/jit_lto/RegisterKernelFragment.h>

__attribute__((__constructor__)) static void
register_interleaved_scan_kernel_4_1_false_true_sc_i_l_b_inner_1_compose()
{
  registerAlgorithm<signed char,
                    int,
                    long,
                    cuvs::neighbors::filtering::ivf_to_sample_filter<
                      long,
                      cuvs::neighbors::filtering::bitset_filter<unsigned int, long>>,
                    cuvs::neighbors::ivf_flat::detail::inner_prod_dist<1, signed char, int>,
                    raft::compose_op<raft::plug_const_op<float, raft::add_op>,
                                     raft::plug_const_op<float, raft::mul_op>>>(
    "interleaved_scan_kernel_4_1_false_true",
    embedded_interleaved_scan_kernel_4_1_false_true_sc_i_l_b_inner_1_compose,
    sizeof(embedded_interleaved_scan_kernel_4_1_false_true_sc_i_l_b_inner_1_compose));
}

#endif

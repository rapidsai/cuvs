#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <cuvs/neighbors/ivf_pq.hpp>

#include "detail/ann_utils.cuh"

#include <raft/core/operators.hpp>
#include <raft/linalg/map.cuh>
#include <raft/linalg/reduce.cuh>

#include <raft/util/cudart_utils.hpp>

namespace cuvs::neighbors::ivf_pq {
index_params index_params::from_dataset(raft::matrix_extent<int64_t> dataset,
                                        cuvs::distance::DistanceType metric)
{
  index_params params;
  params.n_lists =
    dataset.extent(0) < 4 * 2500 ? 4 : static_cast<uint32_t>(std::sqrt(dataset.extent(0)));
  params.n_lists = std::min<uint32_t>(params.n_lists, dataset.extent(0));
  params.pq_dim =
    raft::round_up_safe(static_cast<uint32_t>(dataset.extent(1) / 4), static_cast<uint32_t>(8));
  if (params.pq_dim == 0) params.pq_dim = 8;
  params.pq_bits                  = 8;
  params.kmeans_trainset_fraction = dataset.extent(0) < 10000 ? 1 : 0.1;
  params.metric                   = metric;
  return params;
}

template <typename IdxT>
index<IdxT>::index(raft::resources const& handle)
  // this constructor is just for a temporary index, for use in the deserialization
  // api. all the parameters here will get replaced with loaded values - that aren't
  // necessarily known ahead of time before deserialization.
  // TODO: do we even need a handle here - could just construct one?
  : index(handle,
          cuvs::distance::DistanceType::L2Expanded,
          codebook_gen::PER_SUBSPACE,
          0,
          0,
          8,
          0,
          true)
{
}

template <typename IdxT>
index<IdxT>::index(raft::resources const& handle, const index_params& params, uint32_t dim)
  : index(handle,
          params.metric,
          params.codebook_kind,
          params.n_lists,
          dim,
          params.pq_bits,
          params.pq_dim,
          params.conservative_memory_allocation)
{
}

template <typename IdxT>
index<IdxT>::index(raft::resources const& handle,
                   cuvs::distance::DistanceType metric,
                   codebook_gen codebook_kind,
                   uint32_t n_lists,
                   uint32_t dim,
                   uint32_t pq_bits,
                   uint32_t pq_dim,
                   bool conservative_memory_allocation)
  : cuvs::neighbors::index(),
    metric_(metric),
    codebook_kind_(codebook_kind),
    dim_(dim),
    pq_bits_(pq_bits),
    pq_dim_(pq_dim == 0 ? calculate_pq_dim(dim) : pq_dim),
    conservative_memory_allocation_(conservative_memory_allocation),
    lists_{n_lists},
    list_sizes_{raft::make_device_vector<uint32_t, uint32_t>(handle, n_lists)},
    pq_centers_{raft::make_device_mdarray<float>(handle, make_pq_centers_extents())},
    centers_{raft::make_device_matrix<float, uint32_t>(handle, n_lists, this->dim_ext())},
    centers_rot_{raft::make_device_matrix<float, uint32_t>(handle, n_lists, this->rot_dim())},
    rotation_matrix_{
      raft::make_device_matrix<float, uint32_t>(handle, this->rot_dim(), this->dim())},
    data_ptrs_{raft::make_device_vector<uint8_t*, uint32_t>(handle, n_lists)},
    inds_ptrs_{raft::make_device_vector<IdxT*, uint32_t>(handle, n_lists)},
    accum_sorted_sizes_{raft::make_host_vector<IdxT, uint32_t>(n_lists + 1)}
{
  check_consistency();
  accum_sorted_sizes_(n_lists) = 0;
}

template <typename IdxT>
IdxT index<IdxT>::size() const noexcept
{
  return accum_sorted_sizes_(n_lists());
}

template <typename IdxT>
uint32_t index<IdxT>::dim() const noexcept
{
  return dim_;
}

template <typename IdxT>
uint32_t index<IdxT>::dim_ext() const noexcept
{
  return raft::round_up_safe(dim() + 1, 8u);
}

template <typename IdxT>
uint32_t index<IdxT>::rot_dim() const noexcept
{
  return pq_len() * pq_dim();
}

template <typename IdxT>
uint32_t index<IdxT>::pq_bits() const noexcept
{
  return pq_bits_;
}

template <typename IdxT>
uint32_t index<IdxT>::pq_dim() const noexcept
{
  return pq_dim_;
}

template <typename IdxT>
uint32_t index<IdxT>::pq_len() const noexcept
{
  return raft::div_rounding_up_unsafe(dim(), pq_dim());
}

template <typename IdxT>
uint32_t index<IdxT>::pq_book_size() const noexcept
{
  return 1 << pq_bits();
}

template <typename IdxT>
cuvs::distance::DistanceType index<IdxT>::metric() const noexcept
{
  return metric_;
}

template <typename IdxT>
codebook_gen index<IdxT>::codebook_kind() const noexcept
{
  return codebook_kind_;
}

template <typename IdxT>
uint32_t index<IdxT>::n_lists() const noexcept
{
  return lists_.size();
}

template <typename IdxT>
bool index<IdxT>::conservative_memory_allocation() const noexcept
{
  return conservative_memory_allocation_;
}

template <typename IdxT>
raft::device_mdspan<float,
                    typename cuvs::neighbors::ivf_pq::index<IdxT>::pq_centers_extents,
                    raft::row_major>
index<IdxT>::pq_centers() noexcept
{
  return pq_centers_.view();
}

template <typename IdxT>
raft::device_mdspan<const float,
                    typename cuvs::neighbors::ivf_pq::index<IdxT>::pq_centers_extents,
                    raft::row_major>
index<IdxT>::pq_centers() const noexcept
{
  return pq_centers_.view();
}

template <typename IdxT>
std::vector<std::shared_ptr<list_data<IdxT>>>& index<IdxT>::lists() noexcept
{
  return lists_;
}

template <typename IdxT>
const std::vector<std::shared_ptr<list_data<IdxT>>>& index<IdxT>::lists() const noexcept
{
  return lists_;
}

template <typename IdxT>
raft::device_vector_view<uint8_t*, uint32_t, raft::row_major> index<IdxT>::data_ptrs() noexcept
{
  return data_ptrs_.view();
}

template <typename IdxT>
raft::device_vector_view<const uint8_t* const, uint32_t, raft::row_major> index<IdxT>::data_ptrs()
  const noexcept
{
  return raft::make_mdspan<const uint8_t* const, uint32_t, raft::row_major, false, true>(
    data_ptrs_.data_handle(), data_ptrs_.extents());
}

template <typename IdxT>
raft::device_vector_view<IdxT*, uint32_t, raft::row_major> index<IdxT>::inds_ptrs() noexcept
{
  return inds_ptrs_.view();
}

template <typename IdxT>
raft::device_vector_view<const IdxT* const, uint32_t, raft::row_major> index<IdxT>::inds_ptrs()
  const noexcept
{
  return raft::make_mdspan<const IdxT* const, uint32_t, raft::row_major, false, true>(
    inds_ptrs_.data_handle(), inds_ptrs_.extents());
}

template <typename IdxT>
raft::device_matrix_view<float, uint32_t, raft::row_major> index<IdxT>::rotation_matrix() noexcept
{
  return rotation_matrix_.view();
}

template <typename IdxT>
raft::device_matrix_view<const float, uint32_t, raft::row_major> index<IdxT>::rotation_matrix()
  const noexcept
{
  return rotation_matrix_.view();
}

template <typename IdxT>
raft::host_vector_view<IdxT, uint32_t, raft::row_major> index<IdxT>::accum_sorted_sizes() noexcept
{
  return accum_sorted_sizes_.view();
}

template <typename IdxT>
raft::host_vector_view<const IdxT, uint32_t, raft::row_major> index<IdxT>::accum_sorted_sizes()
  const noexcept
{
  return accum_sorted_sizes_.view();
}

template <typename IdxT>
raft::device_vector_view<uint32_t, uint32_t, raft::row_major> index<IdxT>::list_sizes() noexcept
{
  return list_sizes_.view();
}

template <typename IdxT>
raft::device_vector_view<const uint32_t, uint32_t, raft::row_major> index<IdxT>::list_sizes()
  const noexcept
{
  return list_sizes_.view();
}

template <typename IdxT>
raft::device_matrix_view<float, uint32_t, raft::row_major> index<IdxT>::centers() noexcept
{
  return centers_.view();
}

template <typename IdxT>
raft::device_matrix_view<const float, uint32_t, raft::row_major> index<IdxT>::centers()
  const noexcept
{
  return centers_.view();
}

template <typename IdxT>
raft::device_matrix_view<float, uint32_t, raft::row_major> index<IdxT>::centers_rot() noexcept
{
  return centers_rot_.view();
}

template <typename IdxT>
raft::device_matrix_view<const float, uint32_t, raft::row_major> index<IdxT>::centers_rot()
  const noexcept
{
  return centers_rot_.view();
}

template <typename IdxT>
uint32_t index<IdxT>::get_list_size_in_bytes(uint32_t label)
{
  RAFT_EXPECTS(label < this->n_lists(),
               "Expected label to be less than number of lists in the index");
  auto& list_data = this->lists()[label]->data;
  return list_data.size();
}

template <typename IdxT>
void index<IdxT>::check_consistency()
{
  RAFT_EXPECTS(pq_bits() >= 4 && pq_bits() <= 8,
               "`pq_bits` must be within closed range [4,8], but got %u.",
               pq_bits());
  RAFT_EXPECTS((pq_bits() * pq_dim()) % 8 == 0,
               "`pq_bits * pq_dim` must be a multiple of 8, but got %u * %u = %u.",
               pq_bits(),
               pq_dim(),
               pq_bits() * pq_dim());
}

template <typename IdxT>
typename index<IdxT>::pq_centers_extents index<IdxT>::make_pq_centers_extents()
{
  switch (codebook_kind()) {
    case codebook_gen::PER_SUBSPACE:
      return raft::make_extents<uint32_t>(pq_dim(), pq_len(), pq_book_size());
    case codebook_gen::PER_CLUSTER:
      return raft::make_extents<uint32_t>(n_lists(), pq_len(), pq_book_size());
    default: RAFT_FAIL("Unreachable code");
  }
}

template <typename IdxT>
uint32_t index<IdxT>::calculate_pq_dim(uint32_t dim)
{
  // If the dimensionality is large enough, we can reduce it to improve performance
  if (dim >= 128) { dim /= 2; }
  // Round it down to 32 to improve performance.
  auto r = raft::round_down_safe<uint32_t>(dim, 32);
  if (r > 0) return r;
  // If the dimensionality is really low, round it to the closest power-of-two
  r = 1;
  while ((r << 1) <= dim) {
    r = r << 1;
  }
  return r;
}

template <typename IdxT>
raft::device_matrix_view<const int8_t, uint32_t, raft::row_major> index<IdxT>::rotation_matrix_int8(
  const raft::resources& res) const
{
  if (!rotation_matrix_int8_.has_value()) {
    rotation_matrix_int8_.emplace(
      raft::make_device_mdarray<int8_t, uint32_t>(res, rotation_matrix().extents()));
    raft::linalg::map(res,
                      rotation_matrix_int8_->view(),
                      cuvs::spatial::knn::detail::utils::mapping<int8_t>{},
                      rotation_matrix());
  }
  return rotation_matrix_int8_->view();
}

template <typename IdxT>
raft::device_matrix_view<const int8_t, uint32_t, raft::row_major> index<IdxT>::centers_int8(
  const raft::resources& res) const
{
  if (!centers_int8_.has_value()) {
    uint32_t n_lists      = this->n_lists();
    uint32_t dim          = this->dim();
    uint32_t dim_ext      = this->dim_ext();
    uint32_t dim_ext_int8 = raft::round_up_safe(dim + 2, 16u);
    centers_int8_.emplace(raft::make_device_matrix<int8_t, uint32_t>(res, n_lists, dim_ext_int8));
    auto* inputs = centers().data_handle();
    /* NOTE: maximizing the range and the precision of int8_t GEMM

    int8_t has a very limited range [-128, 127], which is problematic when storing both vectors and
    their squared norms in one place.

    We map all dimensions by multiplying by 128. But that means we need to multiply the squared norm
    component by `128^2`, which we cannot afford, since it most likely overflows.
    So, a naive mapping would be:
    ```
      [c_1 * 128, c_2, * 128, ...., c_(dim-1) * 128,  n2 * 128 * 128, 0 ... 0]
      • [q_1 * 128, q_2 * 128, ..., q_(dim-1)*128, -0.5, 0, ... 0]
    ```

    Which is at first can be improved by moving one 128 to the query side:
    ```
      [c_1 * 128, c_2, * 128, ...., c_(dim-1) * 128,  n2 * 128, 0 ... 0]
      • [q_1 * 128, q_2 * 128, ..., q_(dim-1)*128, -64, 0, ... 0]
    ```

    Yet this still only works for vectors with L2 norms not bigger than one and has a rather awful
    granularity of 64. To improve both the range and the precision, we count the number of available
    slots `m > 2` and decompose the squared norm, such that:
    ```
      0.5 * 128 * n2 = 64 * n2 = 128 * z + (m - 1) * y
    ```
    where `y` maximizes the available range while `z` encodes the rounding error.
    Then we get following dot product during the coarse search:
    ```
      [c_1 * 128, c_2, * 128, ...., c_(dim-1) * 128,  z, y, ... y]
      • [q_1 * 128, q_2 * 128, ..., q_(dim-1)*128, 1 - m,  -128, ... -128]
    ```
    `m` is maximum 16, so we get the coefficient much lower than the naive 64 on the query side; and
    it is limited by the range we can cover (the squared norm must be within `m * 2` before
    normalization).
    */
    raft::linalg::map_offset(
      res, centers_int8_->view(), [dim, dim_ext, dim_ext_int8, inputs] __device__(uint32_t ix) {
        uint32_t col = ix % dim_ext_int8;
        uint32_t row = ix / dim_ext_int8;
        if (col < dim) {
          return static_cast<int8_t>(
            std::clamp(inputs[col + row * dim_ext] * 128.0f, -128.0f, 127.f));
        }
        auto x = inputs[row * dim_ext + dim];
        auto c = 64.0f / static_cast<float>(dim_ext_int8 - dim - 1);
        auto y = std::clamp(x * c, -128.0f, 127.f);
        auto z = std::clamp((y - std::round(y)) * 128.0f, -128.0f, 127.f);
        if (col > dim) { return static_cast<int8_t>(std::round(y)); }
        return static_cast<int8_t>(z);
      });
  }
  return centers_int8_->view();
}

template <typename IdxT>
raft::device_matrix_view<const half, uint32_t, raft::row_major> index<IdxT>::rotation_matrix_half(
  const raft::resources& res) const
{
  if (!rotation_matrix_half_.has_value()) {
    rotation_matrix_half_.emplace(
      raft::make_device_mdarray<half, uint32_t>(res, rotation_matrix().extents()));
    raft::linalg::map(res, rotation_matrix_half_->view(), raft::cast_op<half>{}, rotation_matrix());
  }
  return rotation_matrix_half_->view();
}

template <typename IdxT>
raft::device_matrix_view<const half, uint32_t, raft::row_major> index<IdxT>::centers_half(
  const raft::resources& res) const
{
  if (!centers_half_.has_value()) {
    centers_half_.emplace(raft::make_device_mdarray<half, uint32_t>(res, centers().extents()));
    raft::linalg::map(res, centers_half_->view(), raft::cast_op<half>{}, centers());
  }
  return centers_half_->view();
}

template struct index<int64_t>;

}  // namespace cuvs::neighbors::ivf_pq

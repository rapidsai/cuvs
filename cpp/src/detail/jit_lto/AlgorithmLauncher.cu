/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuvs/detail/jit_lto/AlgorithmLauncher.h>

#include <raft/util/cuda_rt_essentials.hpp>

AlgorithmLauncher::AlgorithmLauncher(hipKernel_t k) : kernel{k} {}

void AlgorithmLauncher::call(
  hipStream_t stream, dim3 grid, dim3 block, std::size_t shared_mem, void** kernel_args)
{
  hipLaunchAttribute attribute[1];
  attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
  attribute[0].val.programmaticStreamSerializationAllowed = 1;

  hipLaunchConfig_t config;
  config.gridDim          = grid;
  config.blockDim         = block;
  config.stream           = stream;
  config.attrs            = attribute;
  config.numAttrs         = 1;
  config.dynamicSmemBytes = shared_mem;

  RAFT_CUDA_TRY(hipLaunchKernelExC(&config, kernel, kernel_args));
}

std::unordered_map<std::string, std::shared_ptr<AlgorithmLauncher>>& get_cached_launchers()
{
  static std::unordered_map<std::string, std::shared_ptr<AlgorithmLauncher>> launchers;
  return launchers;
}

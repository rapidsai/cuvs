/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuvs/detail/jit_lto/AlgorithmLauncher.h>

#include <iostream>

AlgorithmLauncher::AlgorithmLauncher(hipLibrary_t l, hipKernel_t k) : library{l}, kernel{k} {}

void AlgorithmLauncher::call(
  hipStream_t stream, dim3 grid, dim3 block, std::size_t shared_mem, void** kernel_args)
{
  hipLaunchAttribute attribute[1];
  attribute[0].id = CU_LAUNCH_ATTRIBUTE_PROGRAMMATIC_STREAM_SERIALIZATION;
  attribute[0].value.programmaticStreamSerializationAllowed = 1;

  HIP_LAUNCH_CONFIG config{};
  config.gridDimX       = grid.x;
  config.gridDimY       = grid.y;
  config.gridDimZ       = grid.z;
  config.blockDimX      = block.x;
  config.blockDimY      = block.y;
  config.blockDimZ      = block.z;
  config.sharedMemBytes = shared_mem;
  config.hStream        = stream;
  config.attrs          = attribute;
  config.numAttrs       = 1;

  hipDrvLaunchKernelEx(&config, (hipFunction_t)kernel, kernel_args, 0);
}

std::unordered_map<std::string, AlgorithmLauncher>& get_cached_launchers()
{
  static std::unordered_map<std::string, AlgorithmLauncher> launchers;
  return launchers;
}

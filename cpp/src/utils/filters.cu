#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuvs/core/bitmap.hpp>
#include <cuvs/core/bitset.hpp>
#include <faiss/impl/IDSelector.h>
#include <raft/core/bitset.cuh>

namespace cuvs::utils {

/**
 * @brief CUDA kernel to set a range of bits in a bitset to true
 *
 * @param bitset_data Pointer to the bitset data
 * @param imin Starting index
 * @param imax Ending index
 * @param n_elements_to_set Number of elements to set
 */
template <typename bitset_t>
RAFT_KERNEL set_range_kernel(bitset_t* bitset_data,
                             uint32_t imin,
                             uint32_t imax,
                             uint32_t n_elements_to_set)
{
  uint32_t idx         = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t nbits = sizeof(bitset_t) * 8;

  uint32_t current_index = (imin / nbits) + idx;
  bitset_t mask          = 0;
  if (idx < n_elements_to_set) {
    if (n_elements_to_set == 1) {
      // Special case: range is within a single element
      int bit_offset = imin % nbits;
      mask           = (bitset_t{1} << bit_offset) - 1;
      bit_offset     = imax % nbits;
      mask           = mask ^ ((bitset_t{1} << bit_offset) - 1);
    } else if (idx == 0) {
      // First element: set bits from imin to end
      int bit_offset = imin % nbits;
      mask           = ~((bitset_t{1} << bit_offset) - 1);
    } else if (idx == n_elements_to_set - 1) {
      // Last element: set bits from start to imax
      int bit_offset = imax % nbits;
      mask           = (bitset_t{1} << bit_offset) - 1;
    } else {
      // Middle elements: set all bits
      mask = ~mask;
    }
    atomicOr(&bitset_data[current_index], mask);
  }
}

/**
 * @brief Convert a Faiss IDSelectorRange to a cuvs::core::bitset_view
 *
 * @param selector The Faiss IDSelectorRange to convert
 * @param bitset The cuvs::core::bitset_view to store the result
 */
void convert_to_bitset(raft::resources const& res,
                       const faiss::IDSelectorRange& selector,
                       cuvs::core::bitset_view<uint32_t, uint32_t> bitset)
{
  RAFT_EXPECTS(bitset.size() >= selector.imax,
               "IDSelectorRange is out of range for the given bitset");
  const uint32_t nbits = sizeof(uint32_t) * 8;
  auto original_nbits  = bitset.get_original_nbits();
  if (original_nbits == 0) { original_nbits = nbits; }
  uint32_t imin = selector.imin;
  uint32_t imax = selector.imax;

  uint32_t n_elements_to_set = 1 + (imax + original_nbits) / original_nbits;
  n_elements_to_set -= (imin + original_nbits) / original_nbits;
  auto stream = raft::resource::get_cuda_stream(res);

  const int threads_per_block = 256;
  const int blocks            = (n_elements_to_set + threads_per_block - 1) / threads_per_block;

  if (nbits == original_nbits) {
    set_range_kernel<uint32_t><<<blocks, threads_per_block, 0, stream>>>(
      (uint32_t*)bitset.data(), imin, imax, n_elements_to_set);
  } else if (original_nbits == 8) {
    set_range_kernel<uint8_t><<<blocks, threads_per_block, 0, stream>>>(
      (uint8_t*)bitset.data(), imin, imax, n_elements_to_set);
  } else if (original_nbits == 64) {
    set_range_kernel<uint64_t><<<blocks, threads_per_block, 0, stream>>>(
      (uint64_t*)bitset.data(), imin, imax, n_elements_to_set);
  } else {
    throw std::invalid_argument("Unsupported original_nbits");
  }
}

/**
 * @brief Convert a Faiss IDSelector to a cuvs::core::bitset_view
 *
 * @param selector The Faiss IDSelector to convert
 * @param bitset The cuvs::core::bitset_view to store the result
 */
void convert_to_bitset(raft::resources const& res,
                       const faiss::IDSelector& selector,
                       cuvs::core::bitset_view<uint32_t, uint32_t> bitset)
{
}

/**
 * @brief Convert a Faiss IDSelector to a cuvs::core::bitmap
 *
 * @param selector The Faiss IDSelector to convert
 * @param bitmap The cuvs::core::bitmap to store the result
 */
void convert_to_bitmap(raft::resources const& res,
                       const faiss::IDSelector& selector,
                       cuvs::core::bitmap_view<uint32_t, uint32_t> bitmap)
{
}

}  // namespace cuvs::utils

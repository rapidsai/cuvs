/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../common/ann_types.hpp"
#include "cuvs_ann_bench_param_parser.h"

#include <rmm/mr/device/per_device_resource.hpp>

#include <algorithm>
#include <cmath>
#include <memory>
#include <stdexcept>
#include <string>
#include <type_traits>
#include <utility>

namespace cuvs::bench {

#ifdef CUVS_ANN_BENCH_USE_CUVS_MG
void add_distribution_mode(cuvs::neighbors::mg::distribution_mode* dist_mode,
                           const nlohmann::json& conf)
{
  if (conf.contains("distribution_mode")) {
    std::string distribution_mode = conf.at("distribution_mode");
    if (distribution_mode == "replicated") {
      *dist_mode = cuvs::neighbors::mg::distribution_mode::REPLICATED;
    } else if (distribution_mode == "sharded") {
      *dist_mode = cuvs::neighbors::mg::distribution_mode::SHARDED;
    } else {
      throw std::runtime_error("invalid value for distribution_mode");
    }
  } else {
    // default
    *dist_mode = cuvs::neighbors::mg::distribution_mode::SHARDED;
  }
};

void add_merge_mode(cuvs::neighbors::mg::sharded_merge_mode* merge_mode, const nlohmann::json& conf)
{
  if (conf.contains("merge_mode")) {
    std::string sharded_merge_mode = conf.at("merge_mode");
    if (sharded_merge_mode == "tree_merge") {
      *merge_mode = cuvs::neighbors::mg::sharded_merge_mode::TREE_MERGE;
    } else if (sharded_merge_mode == "merge_on_root_rank") {
      *merge_mode = cuvs::neighbors::mg::sharded_merge_mode::MERGE_ON_ROOT_RANK;
    } else {
      throw std::runtime_error("invalid value for merge_mode");
    }
  } else {
    // default
    *merge_mode = cuvs::neighbors::mg::sharded_merge_mode::TREE_MERGE;
  }
};
#endif

template <typename T>
auto create_algo(const std::string& algo_name,
                 const std::string& distance,
                 int dim,
                 const nlohmann::json& conf) -> std::unique_ptr<cuvs::bench::algo<T>>
{
  [[maybe_unused]] cuvs::bench::Metric metric = parse_metric(distance);
  std::unique_ptr<cuvs::bench::algo<T>> a;

  if constexpr (std::is_same_v<T, float>) {
#ifdef CUVS_ANN_BENCH_USE_CUVS_BRUTE_FORCE
    if (algo_name == "raft_brute_force" || algo_name == "cuvs_brute_force") {
      a = std::make_unique<cuvs::bench::cuvs_gpu<T>>(metric, dim);
    }
#endif
  }

  if constexpr (std::is_same_v<T, uint8_t>) {}

#ifdef CUVS_ANN_BENCH_USE_CUVS_IVF_FLAT
  if constexpr (std::is_same_v<T, float> || std::is_same_v<T, uint8_t> ||
                std::is_same_v<T, int8_t>) {
    if (algo_name == "raft_ivf_flat" || algo_name == "cuvs_ivf_flat") {
      typename cuvs::bench::cuvs_ivf_flat<T, int64_t>::build_param param;
      parse_build_param<T, int64_t>(conf, param);
      a = std::make_unique<cuvs::bench::cuvs_ivf_flat<T, int64_t>>(metric, dim, param);
    }
  }
#endif
#ifdef CUVS_ANN_BENCH_USE_CUVS_IVF_PQ
  if (algo_name == "raft_ivf_pq" || algo_name == "cuvs_ivf_pq") {
    typename cuvs::bench::cuvs_ivf_pq<T, int64_t>::build_param param;
    parse_build_param<T, int64_t>(conf, param);
    a = std::make_unique<cuvs::bench::cuvs_ivf_pq<T, int64_t>>(metric, dim, param);
  }
#endif
#ifdef CUVS_ANN_BENCH_USE_CUVS_CAGRA
  if (algo_name == "raft_cagra" || algo_name == "cuvs_cagra") {
    typename cuvs::bench::cuvs_cagra<T, uint32_t>::build_param param;
    parse_build_param<T, uint32_t>(conf, param);
    a = std::make_unique<cuvs::bench::cuvs_cagra<T, uint32_t>>(metric, dim, param);
  }
#endif
#ifdef CUVS_ANN_BENCH_USE_CUVS_MG
  if constexpr (std::is_same_v<T, float> || std::is_same_v<T, uint8_t> ||
                std::is_same_v<T, int8_t>) {
    if (algo_name == "raft_mg_ivf_flat" || algo_name == "cuvs_mg_ivf_flat") {
      typename cuvs::bench::cuvs_mg_ivf_flat<T, int64_t>::build_param param;
      parse_build_param<T, int64_t>(conf, param);
      add_distribution_mode(&param.mode, conf);
      a = std::make_unique<cuvs::bench::cuvs_mg_ivf_flat<T, int64_t>>(metric, dim, param);
    }
  }

  if (algo_name == "raft_mg_ivf_pq" || algo_name == "cuvs_mg_ivf_pq") {
    typename cuvs::bench::cuvs_mg_ivf_pq<T, int64_t>::build_param param;
    parse_build_param<T, int64_t>(conf, param);
    add_distribution_mode(&param.mode, conf);
    a = std::make_unique<cuvs::bench::cuvs_mg_ivf_pq<T, int64_t>>(metric, dim, param);
  }

  if (algo_name == "raft_mg_cagra" || algo_name == "cuvs_mg_cagra") {
    typename cuvs::bench::cuvs_mg_cagra<T, uint32_t>::build_param param;
    parse_build_param<T, uint32_t>(conf, param);
    add_distribution_mode(&param.mode, conf);
    a = std::make_unique<cuvs::bench::cuvs_mg_cagra<T, uint32_t>>(metric, dim, param);
  }

#endif

  if (!a) { throw std::runtime_error("invalid algo: '" + algo_name + "'"); }

  return a;
}

template <typename T>
auto create_search_param(const std::string& algo_name, const nlohmann::json& conf)
  -> std::unique_ptr<typename cuvs::bench::algo<T>::search_param>
{
#ifdef CUVS_ANN_BENCH_USE_CUVS_BRUTE_FORCE
  if (algo_name == "raft_brute_force" || algo_name == "cuvs_brute_force") {
    auto param = std::make_unique<typename cuvs::bench::cuvs_gpu<T>::search_param>();
    return param;
  }
#endif
#ifdef CUVS_ANN_BENCH_USE_CUVS_IVF_FLAT
  if constexpr (std::is_same_v<T, float> || std::is_same_v<T, uint8_t> ||
                std::is_same_v<T, int8_t>) {
    if (algo_name == "raft_ivf_flat" || algo_name == "cuvs_ivf_flat") {
      auto param =
        std::make_unique<typename cuvs::bench::cuvs_ivf_flat<T, int64_t>::search_param>();
      parse_search_param<T, int64_t>(conf, *param);
      return param;
    }
  }
#endif
#ifdef CUVS_ANN_BENCH_USE_CUVS_IVF_PQ
  if (algo_name == "raft_ivf_pq" || algo_name == "cuvs_ivf_pq") {
    auto param = std::make_unique<typename cuvs::bench::cuvs_ivf_pq<T, int64_t>::search_param>();
    parse_search_param<T, int64_t>(conf, *param);
    return param;
  }
#endif
#ifdef CUVS_ANN_BENCH_USE_CUVS_CAGRA
  if (algo_name == "raft_cagra" || algo_name == "cuvs_cagra") {
    auto param = std::make_unique<typename cuvs::bench::cuvs_cagra<T, uint32_t>::search_param>();
    parse_search_param<T, uint32_t>(conf, *param);
    return param;
  }
#endif
#ifdef CUVS_ANN_BENCH_USE_CUVS_MG
  if constexpr (std::is_same_v<T, float> || std::is_same_v<T, uint8_t> ||
                std::is_same_v<T, int8_t>) {
    if (algo_name == "raft_mg_ivf_flat" || algo_name == "cuvs_mg_ivf_flat") {
      auto param =
        std::make_unique<typename cuvs::bench::cuvs_mg_ivf_flat<T, int64_t>::search_param>();
      parse_search_param<T, int64_t>(conf, *param);
      add_merge_mode(&param->merge_mode, conf);
      return param;
    }
  }

  if (algo_name == "raft_mg_ivf_pq" || algo_name == "cuvs_mg_ivf_pq") {
    auto param = std::make_unique<typename cuvs::bench::cuvs_mg_ivf_pq<T, int64_t>::search_param>();
    parse_search_param<T, int64_t>(conf, *param);
    add_merge_mode(&param->merge_mode, conf);
    return param;
  }

  if (algo_name == "raft_mg_cagra" || algo_name == "cuvs_mg_cagra") {
    auto param = std::make_unique<typename cuvs::bench::cuvs_mg_cagra<T, uint32_t>::search_param>();
    parse_search_param<T, uint32_t>(conf, *param);
    add_merge_mode(&param->merge_mode, conf);
    return param;
  }
#endif

  // else
  throw std::runtime_error("invalid algo: '" + algo_name + "'");
}

};  // namespace cuvs::bench

REGISTER_ALGO_INSTANCE(float);
REGISTER_ALGO_INSTANCE(half);
REGISTER_ALGO_INSTANCE(std::int8_t);
REGISTER_ALGO_INSTANCE(std::uint8_t);

#ifdef ANN_BENCH_BUILD_MAIN
#include "../common/benchmark.hpp"
int main(int argc, char** argv) { return cuvs::bench::run_main(argc, argv); }
#endif

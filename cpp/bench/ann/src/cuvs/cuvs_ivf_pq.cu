/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "cuvs_ivf_pq_wrapper.h"

namespace cuvs::bench {
template class cuvs_ivf_pq<float, int64_t>;
// template class cuvs_ivf_pq<half, int64_t>;
template class cuvs_ivf_pq<uint8_t, int64_t>;
template class cuvs_ivf_pq<int8_t, int64_t>;
}  // namespace cuvs::bench

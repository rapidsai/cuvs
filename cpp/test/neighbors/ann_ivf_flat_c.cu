/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/handle.hpp>
#include <raft/random/rng.cuh>

#include "ann_utils.cuh"
#include <cuvs/neighbors/ivf_flat.h>

extern "C" void run_ivf_flat(int64_t n_rows,
                             int64_t n_queries,
                             int64_t n_dim,
                             uint32_t n_neighbors,
                             float* index_data,
                             float* query_data,
                             float* distances_data,
                             int64_t* neighbors_data,
                             cuvsDistanceType metric,
                             size_t n_probes,
                             size_t n_lists);

template <typename T>
void generate_random_data(T* devPtr, size_t size)
{
  raft::handle_t handle;
  raft::random::RngState r(1234ULL);
  raft::random::uniform(handle, r, devPtr, size, T(0.1), T(2.0));
};

template <typename T, typename IdxT>
void recall_eval(T* query_data,
                 T* index_data,
                 IdxT* neighbors,
                 T* distances,
                 size_t n_queries,
                 size_t n_rows,
                 size_t n_dim,
                 size_t n_neighbors,
                 cuvsDistanceType metric,
                 size_t n_probes,
                 size_t n_lists)
{
  raft::handle_t handle;
  auto distances_ref = raft::make_device_matrix<T, IdxT>(handle, n_queries, n_neighbors);
  auto neighbors_ref = raft::make_device_matrix<IdxT, IdxT>(handle, n_queries, n_neighbors);
  cuvs::neighbors::naive_knn<T, T, IdxT>(
    handle,
    distances_ref.data_handle(),
    neighbors_ref.data_handle(),
    query_data,
    index_data,
    n_queries,
    n_rows,
    n_dim,
    n_neighbors,
    static_cast<cuvs::distance::DistanceType>((uint16_t)metric));

  size_t size = n_queries * n_neighbors;
  std::vector<IdxT> neighbors_h(size);
  std::vector<T> distances_h(size);
  std::vector<IdxT> neighbors_ref_h(size);
  std::vector<T> distances_ref_h(size);

  auto stream = raft::resource::get_cuda_stream(handle);
  raft::copy(neighbors_h.data(), neighbors, size, stream);
  raft::copy(distances_h.data(), distances, size, stream);
  raft::copy(neighbors_ref_h.data(), neighbors_ref.data_handle(), size, stream);
  raft::copy(distances_ref_h.data(), distances_ref.data_handle(), size, stream);

  // verify output
  double min_recall = static_cast<double>(n_probes) / static_cast<double>(n_lists);
  ASSERT_TRUE(cuvs::neighbors::eval_neighbours(neighbors_ref_h,
                                               neighbors_h,
                                               distances_ref_h,
                                               distances_h,
                                               n_queries,
                                               n_neighbors,
                                               0.001,
                                               min_recall));
};

TEST(IvfFlatC, BuildSearch)
{
  int64_t n_rows       = 8096;
  int64_t n_queries    = 128;
  int64_t n_dim        = 32;
  uint32_t n_neighbors = 8;

  raft::handle_t handle;
  auto stream = raft::resource::get_cuda_stream(handle);

  cuvsDistanceType metric = L2Expanded;
  size_t n_probes         = 20;
  size_t n_lists          = 1024;

  rmm::device_uvector<float> index_data(n_rows * n_dim, stream);
  rmm::device_uvector<float> query_data(n_queries * n_dim, stream);
  rmm::device_uvector<int64_t> neighbors_data(n_queries * n_neighbors, stream);
  rmm::device_uvector<float> distances_data(n_queries * n_neighbors, stream);

  generate_random_data(index_data.data(), n_rows * n_dim);
  generate_random_data(query_data.data(), n_queries * n_dim);

  run_ivf_flat(n_rows,
               n_queries,
               n_dim,
               n_neighbors,
               index_data.data(),
               query_data.data(),
               distances_data.data(),
               neighbors_data.data(),
               metric,
               n_probes,
               n_lists);

  recall_eval(query_data.data(),
              index_data.data(),
              neighbors_data.data(),
              distances_data.data(),
              n_queries,
              n_rows,
              n_dim,
              n_neighbors,
              metric,
              n_probes,
              n_lists);
}

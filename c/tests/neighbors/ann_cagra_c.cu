/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-2025, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */

#include "test_utils.cuh"
#include <cstddef>
#include <cuvs/core/c_api.h>
#include <cuvs/distance/distance.hpp>
#include <dlpack/dlpack.h>

#include <cstdint>
#include <cstring>
#include <cuvs/neighbors/cagra.h>
#include <cuvs/neighbors/hnsw.h>

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_mdspan.hpp>
#include <raft/core/math.hpp>
#include <raft/core/mdspan.hpp>
#include <raft/core/operators.hpp>
#include <raft/core/resources.hpp>
#include <raft/matrix/argmin.cuh>
#include <raft/matrix/linewise_op.cuh>
#include <sys/types.h>

#include <raft/random/make_blobs.cuh>

float dataset[4][2] = {{0.74021935, 0.9209938},
                       {0.03902049, 0.9689629},
                       {0.92514056, 0.4463501},
                       {0.6673192, 0.10993068}};
float queries[4][2] = {{0.48216683, 0.0428398},
                       {0.5084142, 0.6545497},
                       {0.51260436, 0.2643005},
                       {0.05198065, 0.5789965}};

uint32_t filter[1] = {0b1001};  // index 1 and 2 are removed

uint32_t neighbors_exp[4] = {3, 0, 3, 1};
float distances_exp[4]    = {0.03878258, 0.12472608, 0.04776672, 0.15224178};

uint32_t neighbors_exp_filtered[4] = {3, 0, 3, 0};
float distances_exp_filtered[4]    = {0.03878258, 0.12472608, 0.04776672, 0.59063464};

std::vector<uint64_t> neighbors_exp_disk = {3, 0, 3, 1};
std::vector<float> distances_exp_disk    = {0.03878258, 0.12472608, 0.04776672, 0.15224178};

TEST(CagraC, BuildSearch)
{
  // create cuvsResources_t
  cuvsResources_t res;
  cuvsResourcesCreate(&res);
  hipStream_t stream;
  cuvsStreamGet(res, &stream);

  // create dataset DLTensor
  DLManagedTensor dataset_tensor;
  dataset_tensor.dl_tensor.data               = dataset;
  dataset_tensor.dl_tensor.device.device_type = kDLCPU;
  dataset_tensor.dl_tensor.ndim               = 2;
  dataset_tensor.dl_tensor.dtype.code         = kDLFloat;
  dataset_tensor.dl_tensor.dtype.bits         = 32;
  dataset_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t dataset_shape[2]                    = {4, 2};
  dataset_tensor.dl_tensor.shape              = dataset_shape;
  dataset_tensor.dl_tensor.strides            = nullptr;

  // create index
  cuvsCagraIndex_t index;
  cuvsCagraIndexCreate(&index);

  // build index
  cuvsCagraIndexParams_t build_params;
  cuvsCagraIndexParamsCreate(&build_params);
  cuvsCagraBuild(res, build_params, &dataset_tensor, index);

  // create queries DLTensor
  rmm::device_uvector<float> queries_d(4 * 2, stream);
  raft::copy(queries_d.data(), (float*)queries, 4 * 2, stream);

  DLManagedTensor queries_tensor;
  queries_tensor.dl_tensor.data               = queries_d.data();
  queries_tensor.dl_tensor.device.device_type = kDLCUDA;
  queries_tensor.dl_tensor.ndim               = 2;
  queries_tensor.dl_tensor.dtype.code         = kDLFloat;
  queries_tensor.dl_tensor.dtype.bits         = 32;
  queries_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t queries_shape[2]                    = {4, 2};
  queries_tensor.dl_tensor.shape              = queries_shape;
  queries_tensor.dl_tensor.strides            = nullptr;

  // create neighbors DLTensor
  rmm::device_uvector<uint32_t> neighbors_d(4, stream);

  DLManagedTensor neighbors_tensor;
  neighbors_tensor.dl_tensor.data               = neighbors_d.data();
  neighbors_tensor.dl_tensor.device.device_type = kDLCUDA;
  neighbors_tensor.dl_tensor.ndim               = 2;
  neighbors_tensor.dl_tensor.dtype.code         = kDLUInt;
  neighbors_tensor.dl_tensor.dtype.bits         = 32;
  neighbors_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t neighbors_shape[2]                    = {4, 1};
  neighbors_tensor.dl_tensor.shape              = neighbors_shape;
  neighbors_tensor.dl_tensor.strides            = nullptr;

  // create distances DLTensor
  rmm::device_uvector<float> distances_d(4, stream);

  DLManagedTensor distances_tensor;
  distances_tensor.dl_tensor.data               = distances_d.data();
  distances_tensor.dl_tensor.device.device_type = kDLCUDA;
  distances_tensor.dl_tensor.ndim               = 2;
  distances_tensor.dl_tensor.dtype.code         = kDLFloat;
  distances_tensor.dl_tensor.dtype.bits         = 32;
  distances_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t distances_shape[2]                    = {4, 1};
  distances_tensor.dl_tensor.shape              = distances_shape;
  distances_tensor.dl_tensor.strides            = nullptr;

  cuvsFilter filter;
  filter.type = NO_FILTER;
  filter.addr = (uintptr_t)NULL;

  // search index
  cuvsCagraSearchParams_t search_params;
  cuvsCagraSearchParamsCreate(&search_params);
  cuvsCagraSearch(
    res, search_params, index, &queries_tensor, &neighbors_tensor, &distances_tensor, filter);

  // verify output
  ASSERT_TRUE(
    cuvs::devArrMatchHost(neighbors_exp, neighbors_d.data(), 4, cuvs::Compare<uint32_t>()));
  ASSERT_TRUE(cuvs::devArrMatchHost(
    distances_exp, distances_d.data(), 4, cuvs::CompareApprox<float>(0.001f)));

  // de-allocate index and res
  cuvsCagraSearchParamsDestroy(search_params);
  cuvsCagraIndexParamsDestroy(build_params);
  cuvsCagraIndexDestroy(index);
  cuvsResourcesDestroy(res);
}

TEST(CagraC, BuildExtendSearch)
{
  // create cuvsResources_t
  cuvsResources_t res;
  cuvsResourcesCreate(&res);
  hipStream_t stream;
  cuvsStreamGet(res, &stream);

  raft::resources handle;

  const int32_t dimensions = 16;
  // main_data_size needs to be >= 128 (see issue #486)
  const int32_t main_data_size       = 1024;
  const int32_t additional_data_size = 64;
  const int32_t num_queries          = 4;

  // create random data for datasets and queries
  rmm::device_uvector<float> random_data_d(
    (main_data_size + additional_data_size + num_queries) * dimensions, stream);
  rmm::device_uvector<int32_t> random_labels_d(
    (main_data_size + additional_data_size + num_queries) * dimensions, stream);
  raft::random::make_blobs(random_data_d.data(),
                           random_labels_d.data(),
                           main_data_size + additional_data_size + num_queries,
                           dimensions,
                           10,
                           stream);

  // create  dataset DLTensor
  rmm::device_uvector<float> main_d(main_data_size * dimensions, stream);
  rmm::device_uvector<int32_t> main_labels_d(main_data_size, stream);
  raft::copy(main_d.data(), random_data_d.data(), main_data_size * dimensions, stream);
  DLManagedTensor dataset_tensor;
  dataset_tensor.dl_tensor.data               = main_d.data();
  dataset_tensor.dl_tensor.device.device_type = kDLCUDA;
  dataset_tensor.dl_tensor.ndim               = 2;
  dataset_tensor.dl_tensor.dtype.code         = kDLFloat;
  dataset_tensor.dl_tensor.dtype.bits         = 32;
  dataset_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t dataset_shape[2]                    = {main_data_size, dimensions};
  dataset_tensor.dl_tensor.shape              = dataset_shape;
  dataset_tensor.dl_tensor.strides            = nullptr;

  // create additional dataset DLTensor
  rmm::device_uvector<float> additional_d(additional_data_size * dimensions, stream);
  raft::copy(additional_d.data(),
             random_data_d.data() + main_d.size(),
             additional_data_size * dimensions,
             stream);
  DLManagedTensor additional_dataset_tensor;
  additional_dataset_tensor.dl_tensor.data               = additional_d.data();
  additional_dataset_tensor.dl_tensor.device.device_type = kDLCUDA;
  additional_dataset_tensor.dl_tensor.ndim               = 2;
  additional_dataset_tensor.dl_tensor.dtype.code         = kDLFloat;
  additional_dataset_tensor.dl_tensor.dtype.bits         = 32;
  additional_dataset_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t additional_dataset_shape[2]                    = {additional_data_size, dimensions};
  additional_dataset_tensor.dl_tensor.shape              = additional_dataset_shape;
  additional_dataset_tensor.dl_tensor.strides            = nullptr;

  // create index
  cuvsCagraIndex_t index;
  cuvsCagraIndexCreate(&index);

  // build index
  cuvsCagraIndexParams_t build_params;
  cuvsCagraIndexParamsCreate(&build_params);
  cuvsCagraBuild(res, build_params, &dataset_tensor, index);

  cuvsStreamSync(res);

  // extend index
  cuvsCagraExtendParams_t extend_params;
  cuvsCagraExtendParamsCreate(&extend_params);
  cuvsCagraExtend(res, extend_params, &additional_dataset_tensor, index);

  // create queries DLTensor
  rmm::device_uvector<float> queries_d(num_queries * dimensions, stream);
  raft::copy(queries_d.data(),
             random_data_d.data() + (main_data_size + additional_data_size) * dimensions,
             num_queries * dimensions,
             stream);
  DLManagedTensor queries_tensor;
  queries_tensor.dl_tensor.data               = queries_d.data();
  queries_tensor.dl_tensor.device.device_type = kDLCUDA;
  queries_tensor.dl_tensor.ndim               = 2;
  queries_tensor.dl_tensor.dtype.code         = kDLFloat;
  queries_tensor.dl_tensor.dtype.bits         = 32;
  queries_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t queries_shape[2]                    = {4, dimensions};
  queries_tensor.dl_tensor.shape              = queries_shape;
  queries_tensor.dl_tensor.strides            = nullptr;

  // create pairwise distance matrix for dataset and queries
  auto pairwise_distance_dataset_input =
    raft::make_device_matrix<float>(handle, main_data_size + additional_data_size, dimensions);

  raft::copy(pairwise_distance_dataset_input.data_handle(), main_d.data(), main_d.size(), stream);
  raft::copy(pairwise_distance_dataset_input.data_handle() + main_d.size(),
             additional_d.data(),
             additional_d.size(),
             stream);

  auto pairwise_distance_queries_input =
    raft::make_device_matrix<float>(handle, num_queries, dimensions);

  raft::copy(pairwise_distance_queries_input.data_handle(),
             (float*)queries_d.data(),
             num_queries * dimensions,
             stream);

  auto pairwise_distances =
    raft::make_device_matrix<float>(handle, num_queries, (main_data_size + additional_data_size));
  auto metric = cuvs::distance::DistanceType::L2Expanded;

  cuvs::distance::pairwise_distance(handle,
                                    pairwise_distance_queries_input.view(),
                                    pairwise_distance_dataset_input.view(),

                                    pairwise_distances.view(),
                                    metric);

  auto min_cols =
    raft::make_device_vector<uint32_t, uint32_t>(handle, pairwise_distances.extent(0));

  auto distances_const_view = raft::make_device_matrix_view<const float, uint32_t>(
    pairwise_distances.data_handle(), pairwise_distances.extent(0), pairwise_distances.extent(1));

  raft::matrix::argmin(handle, distances_const_view, min_cols.view());

  float min_cols_distances[num_queries];

  for (uint32_t i = 0; i < min_cols.extent(0); i++) {
    uint32_t mc           = min_cols(i);
    min_cols_distances[i] = pairwise_distances(i, mc);
  }

  // create neighbors DLTensor
  rmm::device_uvector<uint32_t> neighbors_d(4, stream);

  DLManagedTensor neighbors_tensor;
  neighbors_tensor.dl_tensor.data               = neighbors_d.data();
  neighbors_tensor.dl_tensor.device.device_type = kDLCUDA;
  neighbors_tensor.dl_tensor.ndim               = 2;
  neighbors_tensor.dl_tensor.dtype.code         = kDLUInt;
  neighbors_tensor.dl_tensor.dtype.bits         = 32;
  neighbors_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t neighbors_shape[2]                    = {num_queries, 1};
  neighbors_tensor.dl_tensor.shape              = neighbors_shape;
  neighbors_tensor.dl_tensor.strides            = nullptr;

  // create distances DLTensor
  rmm::device_uvector<float> distances_d(4, stream);

  distances_d.resize(4, stream);

  DLManagedTensor distances_tensor;
  distances_tensor.dl_tensor.data               = distances_d.data();
  distances_tensor.dl_tensor.device.device_type = kDLCUDA;
  distances_tensor.dl_tensor.ndim               = 2;
  distances_tensor.dl_tensor.dtype.code         = kDLFloat;
  distances_tensor.dl_tensor.dtype.bits         = 32;
  distances_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t distances_shape[2]                    = {num_queries, 1};
  distances_tensor.dl_tensor.shape              = distances_shape;
  distances_tensor.dl_tensor.strides            = nullptr;

  cuvsFilter filter;
  filter.type = NO_FILTER;
  filter.addr = (uintptr_t)NULL;

  // search index
  cuvsCagraSearchParams_t search_params;
  cuvsCagraSearchParamsCreate(&search_params);
  cuvsCagraSearch(
    res, search_params, index, &queries_tensor, &neighbors_tensor, &distances_tensor, filter);

  // check neighbors
  ASSERT_TRUE(
    cuvs::devArrMatch(min_cols.data_handle(), neighbors_d.data(), 4, cuvs::Compare<uint32_t>()));

  // check distances
  ASSERT_TRUE(cuvs::devArrMatchHost(
    min_cols_distances, distances_d.data(), 4, cuvs::CompareApprox<float>(0.001f)));

  // de-allocate index and res
  cuvsCagraSearchParamsDestroy(search_params);
  cuvsCagraExtendParamsDestroy(extend_params);
  cuvsCagraIndexParamsDestroy(build_params);
  cuvsCagraIndexDestroy(index);
  cuvsResourcesDestroy(res);
}

TEST(CagraC, BuildSearchFiltered)
{
  // create cuvsResources_t
  cuvsResources_t res;
  cuvsResourcesCreate(&res);
  hipStream_t stream;
  cuvsStreamGet(res, &stream);

  // create dataset DLTensor
  DLManagedTensor dataset_tensor;
  dataset_tensor.dl_tensor.data               = dataset;
  dataset_tensor.dl_tensor.device.device_type = kDLCPU;
  dataset_tensor.dl_tensor.ndim               = 2;
  dataset_tensor.dl_tensor.dtype.code         = kDLFloat;
  dataset_tensor.dl_tensor.dtype.bits         = 32;
  dataset_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t dataset_shape[2]                    = {4, 2};
  dataset_tensor.dl_tensor.shape              = dataset_shape;
  dataset_tensor.dl_tensor.strides            = nullptr;

  // create index
  cuvsCagraIndex_t index;
  cuvsCagraIndexCreate(&index);

  // build index
  cuvsCagraIndexParams_t build_params;
  cuvsCagraIndexParamsCreate(&build_params);
  cuvsCagraBuild(res, build_params, &dataset_tensor, index);

  // create queries DLTensor
  rmm::device_uvector<float> queries_d(4 * 2, stream);
  raft::copy(queries_d.data(), (float*)queries, 4 * 2, stream);

  DLManagedTensor queries_tensor;
  queries_tensor.dl_tensor.data               = queries_d.data();
  queries_tensor.dl_tensor.device.device_type = kDLCUDA;
  queries_tensor.dl_tensor.ndim               = 2;
  queries_tensor.dl_tensor.dtype.code         = kDLFloat;
  queries_tensor.dl_tensor.dtype.bits         = 32;
  queries_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t queries_shape[2]                    = {4, 2};
  queries_tensor.dl_tensor.shape              = queries_shape;
  queries_tensor.dl_tensor.strides            = nullptr;

  // create neighbors DLTensor
  rmm::device_uvector<uint32_t> neighbors_d(4, stream);

  DLManagedTensor neighbors_tensor;
  neighbors_tensor.dl_tensor.data               = neighbors_d.data();
  neighbors_tensor.dl_tensor.device.device_type = kDLCUDA;
  neighbors_tensor.dl_tensor.ndim               = 2;
  neighbors_tensor.dl_tensor.dtype.code         = kDLUInt;
  neighbors_tensor.dl_tensor.dtype.bits         = 32;
  neighbors_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t neighbors_shape[2]                    = {4, 1};
  neighbors_tensor.dl_tensor.shape              = neighbors_shape;
  neighbors_tensor.dl_tensor.strides            = nullptr;

  // create distances DLTensor
  rmm::device_uvector<float> distances_d(4, stream);

  DLManagedTensor distances_tensor;
  distances_tensor.dl_tensor.data               = distances_d.data();
  distances_tensor.dl_tensor.device.device_type = kDLCUDA;
  distances_tensor.dl_tensor.ndim               = 2;
  distances_tensor.dl_tensor.dtype.code         = kDLFloat;
  distances_tensor.dl_tensor.dtype.bits         = 32;
  distances_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t distances_shape[2]                    = {4, 1};
  distances_tensor.dl_tensor.shape              = distances_shape;
  distances_tensor.dl_tensor.strides            = nullptr;

  // create filter DLTensor
  rmm::device_uvector<uint32_t> filter_d(1, stream);
  raft::copy(filter_d.data(), filter, 1, stream);

  cuvsFilter filter;

  DLManagedTensor filter_tensor;
  filter_tensor.dl_tensor.data               = filter_d.data();
  filter_tensor.dl_tensor.device.device_type = kDLCUDA;
  filter_tensor.dl_tensor.ndim               = 1;
  filter_tensor.dl_tensor.dtype.code         = kDLUInt;
  filter_tensor.dl_tensor.dtype.bits         = 32;
  filter_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t filter_shape[1]                    = {1};
  filter_tensor.dl_tensor.shape              = filter_shape;
  filter_tensor.dl_tensor.strides            = nullptr;

  filter.type = BITSET;
  filter.addr = (uintptr_t)&filter_tensor;

  // search index
  cuvsCagraSearchParams_t search_params;
  cuvsCagraSearchParamsCreate(&search_params);
  cuvsCagraSearch(
    res, search_params, index, &queries_tensor, &neighbors_tensor, &distances_tensor, filter);
  // verify output
  ASSERT_TRUE(cuvs::devArrMatchHost(
    neighbors_exp_filtered, neighbors_d.data(), 4, cuvs::Compare<uint32_t>()));
  ASSERT_TRUE(cuvs::devArrMatchHost(
    distances_exp_filtered, distances_d.data(), 4, cuvs::CompareApprox<float>(0.001f)));

  // de-allocate index and res
  cuvsCagraSearchParamsDestroy(search_params);
  cuvsCagraIndexParamsDestroy(build_params);
  cuvsCagraIndexDestroy(index);
  cuvsResourcesDestroy(res);
}

TEST(CagraC, BuildMergeSearch)
{
  cuvsResources_t res;
  cuvsResourcesCreate(&res);
  hipStream_t stream;
  cuvsStreamGet(res, &stream);

  float dataset[7][2] = {{0.74021935f, 0.92099380f},
                         {0.03902049f, 0.96896291f},
                         {0.92514056f, 0.44635010f},
                         {0.12345678f, 0.87654321f},
                         {0.50112233f, 0.33221100f},
                         {0.66731918f, 0.10993068f},
                         {0.77777777f, 0.88888888f}};

  float* main_data_ptr       = &dataset[0][0];
  float* additional_data_ptr = &dataset[4][0];
  float* query_data_ptr      = &dataset[6][0];

  rmm::device_uvector<float> main_d(8, stream);
  rmm::device_uvector<float> additional_d(6, stream);
  rmm::device_uvector<float> queries_d(2, stream);
  raft::copy(main_d.data(), main_data_ptr, 8, stream);
  raft::copy(additional_d.data(), additional_data_ptr, 6, stream);
  raft::copy(queries_d.data(), query_data_ptr, 2, stream);

  DLManagedTensor main_dataset_tensor;
  int64_t main_shape[2]                            = {4, 2};
  main_dataset_tensor.dl_tensor.data               = main_d.data();
  main_dataset_tensor.dl_tensor.device.device_type = kDLCUDA;
  main_dataset_tensor.dl_tensor.device.device_id   = 0;
  main_dataset_tensor.dl_tensor.ndim               = 2;
  main_dataset_tensor.dl_tensor.dtype.code         = kDLFloat;
  main_dataset_tensor.dl_tensor.dtype.bits         = 32;
  main_dataset_tensor.dl_tensor.dtype.lanes        = 1;
  main_dataset_tensor.dl_tensor.shape              = main_shape;
  main_dataset_tensor.dl_tensor.strides            = nullptr;

  DLManagedTensor additional_dataset_tensor = main_dataset_tensor;
  int64_t additional_shape[2]               = {3, 2};
  additional_dataset_tensor.dl_tensor.data  = additional_d.data();
  additional_dataset_tensor.dl_tensor.shape = additional_shape;

  DLManagedTensor query_tensor = main_dataset_tensor;
  int64_t query_shape[2]       = {1, 2};
  query_tensor.dl_tensor.data  = queries_d.data();
  query_tensor.dl_tensor.shape = query_shape;

  cuvsCagraIndexParams_t build_params;
  cuvsCagraIndexParamsCreate(&build_params);
  cuvsCagraIndex_t index_main, index_add;
  cuvsCagraIndexCreate(&index_main);
  cuvsCagraIndexCreate(&index_add);
  ASSERT_EQ(cuvsCagraBuild(res, build_params, &main_dataset_tensor, index_main), CUVS_SUCCESS);
  ASSERT_EQ(cuvsCagraBuild(res, build_params, &additional_dataset_tensor, index_add), CUVS_SUCCESS);

  cuvsCagraMergeParams_t merge_params;
  cuvsCagraMergeParamsCreate(&merge_params);
  cuvsCagraIndex_t index_merged;
  cuvsCagraIndexCreate(&index_merged);

  cuvsCagraIndex_t index_array[2] = {index_main, index_add};
  ASSERT_EQ(cuvsCagraMerge(res, merge_params, index_array, 2, index_merged), CUVS_SUCCESS);

  int64_t merged_dim = -1;
  ASSERT_EQ(cuvsCagraIndexGetDims(index_merged, &merged_dim), CUVS_SUCCESS);
  EXPECT_EQ(merged_dim, 2);

  DLManagedTensor neighbors_tensor, distances_tensor;
  rmm::device_uvector<int64_t> neighbors_d(1, stream);
  rmm::device_uvector<float> distances_d(1, stream);
  int64_t neighbors_shape[2]             = {1, 1};
  int64_t distances_shape[2]             = {1, 1};
  neighbors_tensor.dl_tensor.data        = neighbors_d.data();
  neighbors_tensor.dl_tensor.device      = main_dataset_tensor.dl_tensor.device;
  neighbors_tensor.dl_tensor.ndim        = 2;
  neighbors_tensor.dl_tensor.dtype.code  = kDLInt;
  neighbors_tensor.dl_tensor.dtype.bits  = 64;
  neighbors_tensor.dl_tensor.dtype.lanes = 1;
  neighbors_tensor.dl_tensor.shape       = neighbors_shape;
  neighbors_tensor.dl_tensor.strides     = nullptr;
  distances_tensor.dl_tensor.data        = distances_d.data();
  distances_tensor.dl_tensor.device      = main_dataset_tensor.dl_tensor.device;
  distances_tensor.dl_tensor.ndim        = 2;
  distances_tensor.dl_tensor.dtype.code  = kDLFloat;
  distances_tensor.dl_tensor.dtype.bits  = 32;
  distances_tensor.dl_tensor.dtype.lanes = 1;
  distances_tensor.dl_tensor.shape       = distances_shape;
  distances_tensor.dl_tensor.strides     = nullptr;

  cuvsCagraSearchParams_t search_params;
  cuvsCagraSearchParamsCreate(&search_params);
  (*search_params).itopk_size = 1;

  cuvsFilter filter;
  filter.type = NO_FILTER;
  filter.addr = 0;
  ASSERT_EQ(cuvsCagraSearch(res,
                            search_params,
                            index_merged,
                            &query_tensor,
                            &neighbors_tensor,
                            &distances_tensor,
                            filter),
            CUVS_SUCCESS);

  int64_t neighbor_host = -1;
  float distance_host   = 1.0f;
  raft::copy(&neighbor_host, neighbors_d.data(), 1, stream);
  raft::copy(&distance_host, distances_d.data(), 1, stream);
  hipStreamSynchronize(stream);

  EXPECT_EQ(neighbor_host, 6);
  EXPECT_NEAR(distance_host, 0.0f, 1e-6);

  cuvsCagraSearchParamsDestroy(search_params);
  cuvsCagraMergeParamsDestroy(merge_params);
  cuvsCagraIndexParamsDestroy(build_params);
  cuvsCagraIndexDestroy(index_merged);
  cuvsCagraIndexDestroy(index_add);
  cuvsCagraIndexDestroy(index_main);
  cuvsResourcesDestroy(res);
}

TEST(CagraC, BuildSearchACEMemory)
{
  // create cuvsResources_t
  cuvsResources_t res;
  cuvsResourcesCreate(&res);
  hipStream_t stream;
  cuvsStreamGet(res, &stream);

  // create dataset DLTensor
  DLManagedTensor dataset_tensor;
  dataset_tensor.dl_tensor.data               = dataset;
  dataset_tensor.dl_tensor.device.device_type = kDLCPU;
  dataset_tensor.dl_tensor.ndim               = 2;
  dataset_tensor.dl_tensor.dtype.code         = kDLFloat;
  dataset_tensor.dl_tensor.dtype.bits         = 32;
  dataset_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t dataset_shape[2]                    = {4, 2};
  dataset_tensor.dl_tensor.shape              = dataset_shape;
  dataset_tensor.dl_tensor.strides            = nullptr;

  // create index
  cuvsCagraIndex_t index;
  cuvsCagraIndexCreate(&index);

  // build index with ACE memory mode
  cuvsCagraIndexParams_t build_params;
  cuvsCagraIndexParamsCreate(&build_params);
  build_params->build_algo = ACE;

  cuvsAceParams_t ace_params;
  cuvsAceParamsCreate(&ace_params);
  ace_params->npartitions = 2;
  ace_params->ef_construction = 120;
  ace_params->use_disk = false;

  build_params->graph_build_params = ace_params;
  cuvsCagraBuild(res, build_params, &dataset_tensor, index);

  // create queries DLTensor
  rmm::device_uvector<float> queries_d(4 * 2, stream);
  raft::copy(queries_d.data(), (float*)queries, 4 * 2, stream);

  DLManagedTensor queries_tensor;
  queries_tensor.dl_tensor.data               = queries_d.data();
  queries_tensor.dl_tensor.device.device_type = kDLCUDA;
  queries_tensor.dl_tensor.ndim               = 2;
  queries_tensor.dl_tensor.dtype.code         = kDLFloat;
  queries_tensor.dl_tensor.dtype.bits         = 32;
  queries_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t queries_shape[2]                    = {4, 2};
  queries_tensor.dl_tensor.shape              = queries_shape;
  queries_tensor.dl_tensor.strides            = nullptr;

  // create neighbors DLTensor
  rmm::device_uvector<uint32_t> neighbors_d(4, stream);

  DLManagedTensor neighbors_tensor;
  neighbors_tensor.dl_tensor.data               = neighbors_d.data();
  neighbors_tensor.dl_tensor.device.device_type = kDLCUDA;
  neighbors_tensor.dl_tensor.ndim               = 2;
  neighbors_tensor.dl_tensor.dtype.code         = kDLUInt;
  neighbors_tensor.dl_tensor.dtype.bits         = 32;
  neighbors_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t neighbors_shape[2]                    = {4, 1};
  neighbors_tensor.dl_tensor.shape              = neighbors_shape;
  neighbors_tensor.dl_tensor.strides            = nullptr;

  // create distances DLTensor
  rmm::device_uvector<float> distances_d(4, stream);

  DLManagedTensor distances_tensor;
  distances_tensor.dl_tensor.data               = distances_d.data();
  distances_tensor.dl_tensor.device.device_type = kDLCUDA;
  distances_tensor.dl_tensor.ndim               = 2;
  distances_tensor.dl_tensor.dtype.code         = kDLFloat;
  distances_tensor.dl_tensor.dtype.bits         = 32;
  distances_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t distances_shape[2]                    = {4, 1};
  distances_tensor.dl_tensor.shape              = distances_shape;
  distances_tensor.dl_tensor.strides            = nullptr;

  cuvsFilter filter;
  filter.type = NO_FILTER;
  filter.addr = (uintptr_t)NULL;

  // search index
  cuvsCagraSearchParams_t search_params;
  cuvsCagraSearchParamsCreate(&search_params);
  cuvsCagraSearch(
    res, search_params, index, &queries_tensor, &neighbors_tensor, &distances_tensor, filter);

  // verify output
  ASSERT_TRUE(
    cuvs::devArrMatchHost(neighbors_exp, neighbors_d.data(), 4, cuvs::Compare<uint32_t>()));
  ASSERT_TRUE(cuvs::devArrMatchHost(
    distances_exp, distances_d.data(), 4, cuvs::CompareApprox<float>(0.001f)));

  // de-allocate index and res
  cuvsCagraSearchParamsDestroy(search_params);
  cuvsCagraIndexParamsDestroy(build_params);
  cuvsCagraIndexDestroy(index);
  cuvsResourcesDestroy(res);
}

TEST(CagraC, BuildSearchACEDisk)
{
  // create cuvsResources_t
  cuvsResources_t res;
  cuvsResourcesCreate(&res);

  // create dataset DLTensor
  DLManagedTensor dataset_tensor;
  dataset_tensor.dl_tensor.data               = dataset;
  dataset_tensor.dl_tensor.device.device_type = kDLCPU;
  dataset_tensor.dl_tensor.ndim               = 2;
  dataset_tensor.dl_tensor.dtype.code         = kDLFloat;
  dataset_tensor.dl_tensor.dtype.bits         = 32;
  dataset_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t dataset_shape[2]                    = {4, 2};
  dataset_tensor.dl_tensor.shape              = dataset_shape;
  dataset_tensor.dl_tensor.strides            = nullptr;

  // create index
  cuvsCagraIndex_t index;
  cuvsCagraIndexCreate(&index);

  // build index with ACE memory mode
  cuvsCagraIndexParams_t build_params;
  cuvsCagraIndexParamsCreate(&build_params);
  build_params->build_algo = ACE;

  cuvsAceParams_t ace_params;
  cuvsAceParamsCreate(&ace_params);
  ace_params->npartitions = 2;
  ace_params->ef_construction = 120;
  ace_params->use_disk = true;
  ace_params->build_dir = strdup("/tmp/cagra_ace_test_disk");

  build_params->graph_build_params = ace_params;
  cuvsCagraBuild(res, build_params, &dataset_tensor, index);

  // Verify the index is stored on disk
  bool on_disk = false;
  cuvsCagraIndexIsOnDisk(index, &on_disk);
  ASSERT_TRUE(on_disk);

  // Convert CAGRA index to HNSW (automatically serializes to disk for ACE)
  cuvsHnswIndex_t hnsw_index_ser;
  cuvsHnswIndexCreate(&hnsw_index_ser);
  cuvsHnswIndexParams_t hnsw_params;
  cuvsHnswIndexParamsCreate(&hnsw_params);

  cuvsHnswFromCagra(res, hnsw_params, index, hnsw_index_ser);
  ASSERT_EQ(hnsw_index_ser->addr, 0);
  cuvsHnswIndexDestroy(hnsw_index_ser);

  DLManagedTensor queries_tensor;
  queries_tensor.dl_tensor.data               = queries;
  queries_tensor.dl_tensor.device.device_type = kDLCPU;
  queries_tensor.dl_tensor.ndim               = 2;
  queries_tensor.dl_tensor.dtype.code         = kDLFloat;
  queries_tensor.dl_tensor.dtype.bits         = 32;
  queries_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t queries_shape[2]                    = {4, 2};
  queries_tensor.dl_tensor.shape              = queries_shape;
  queries_tensor.dl_tensor.strides            = nullptr;

  // create neighbors DLTensor
  std::vector<uint64_t> neighbors(4);

  DLManagedTensor neighbors_tensor;
  neighbors_tensor.dl_tensor.data               = neighbors.data();
  neighbors_tensor.dl_tensor.device.device_type = kDLCPU;
  neighbors_tensor.dl_tensor.ndim               = 2;
  neighbors_tensor.dl_tensor.dtype.code         = kDLUInt;
  neighbors_tensor.dl_tensor.dtype.bits         = 64;
  neighbors_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t neighbors_shape[2]                    = {4, 1};
  neighbors_tensor.dl_tensor.shape              = neighbors_shape;
  neighbors_tensor.dl_tensor.strides            = nullptr;

  // create distances DLTensor
  std::vector<float> distances(4);

  DLManagedTensor distances_tensor;
  distances_tensor.dl_tensor.data               = distances.data();
  distances_tensor.dl_tensor.device.device_type = kDLCPU;
  distances_tensor.dl_tensor.ndim               = 2;
  distances_tensor.dl_tensor.dtype.code         = kDLFloat;
  distances_tensor.dl_tensor.dtype.bits         = 32;
  distances_tensor.dl_tensor.dtype.lanes        = 1;
  int64_t distances_shape[2]                    = {4, 1};
  distances_tensor.dl_tensor.shape              = distances_shape;
  distances_tensor.dl_tensor.strides            = nullptr;

  // Deserialize the HNSW index from disk for search
  cuvsHnswIndex_t hnsw_index;
  cuvsHnswIndexCreate(&hnsw_index);
  hnsw_index->dtype = index->dtype;

  // Use the actual dimension from the dataset
  int dim = dataset_tensor.dl_tensor.shape[1];
  cuvsHnswDeserialize(res, hnsw_params, "/tmp/cagra_ace_test_disk/hnsw_index.bin", dim, L2Expanded, hnsw_index);
  ASSERT_NE(hnsw_index->addr, 0);

  // Search the HNSW index
  cuvsHnswSearchParams_t search_params;
  cuvsHnswSearchParamsCreate(&search_params);
  cuvsHnswSearch(
    res, search_params, hnsw_index, &queries_tensor, &neighbors_tensor, &distances_tensor);

  // Verify output
  ASSERT_TRUE(cuvs::hostVecMatch(neighbors_exp_disk, neighbors, cuvs::Compare<uint64_t>()));
  ASSERT_TRUE(cuvs::hostVecMatch(distances_exp_disk, distances, cuvs::CompareApprox<float>(0.001f)));

  cuvsCagraIndexParamsDestroy(build_params);
  cuvsCagraIndexDestroy(index);
  cuvsHnswSearchParamsDestroy(search_params);
  cuvsHnswIndexParamsDestroy(hnsw_params);
  cuvsHnswIndexDestroy(hnsw_index);
  cuvsResourcesDestroy(res);
}

/*
 * SPDX-FileCopyrightText: Copyright (c) 2024, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <hip/hip_runtime.h>

#include <gtest/gtest.h>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/handle.hpp>
#include <raft/random/rng.cuh>

#include <cuvs/distance/distance.h>

extern "C" void run_pairwise_distance(int64_t n_rows,
                                      int64_t n_queries,
                                      int64_t n_dim,
                                      float* index_data,
                                      float* query_data,
                                      float* distances_data,
                                      cuvsDistanceType metric);

template <typename T>
void generate_random_data(T* devPtr, size_t size)
{
  raft::handle_t handle;
  raft::random::RngState r(1234ULL);
  raft::random::uniform(handle, r, devPtr, size, T(0.1), T(2.0));
};

TEST(PairwiseDistanceC, Distance)
{
  int64_t n_rows    = 8096;
  int64_t n_queries = 128;
  int64_t n_dim     = 32;

  cuvsDistanceType metric = L2Expanded;

  float *index_data, *query_data, *distances_data;
  hipMalloc(&index_data, sizeof(float) * n_rows * n_dim);
  hipMalloc(&query_data, sizeof(float) * n_queries * n_dim);
  hipMalloc(&distances_data, sizeof(float) * n_queries * n_rows);

  generate_random_data(index_data, n_rows * n_dim);
  generate_random_data(query_data, n_queries * n_dim);

  run_pairwise_distance(n_rows, n_queries, n_dim, index_data, query_data, distances_data, metric);

  // delete device memory
  hipFree(index_data);
  hipFree(query_data);
  hipFree(distances_data);
}
